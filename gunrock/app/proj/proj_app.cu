// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file proj_app.cu
 *
 * @brief Simple Gunrock Application
 */

#include <gunrock/gunrock.h>
#include <gunrock/util/test_utils.cuh>
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

#include <gunrock/app/proj/proj_enactor.cuh>
#include <gunrock/app/proj/proj_test.cuh>

namespace gunrock {
namespace app {
namespace proj {


hipError_t UseParameters(util::Parameters &parameters)
{
    hipError_t retval = hipSuccess;
    GUARD_CU(UseParameters_app(parameters));
    GUARD_CU(UseParameters_problem(parameters));
    GUARD_CU(UseParameters_enactor(parameters));
    return retval;
}

/**
 * @brief Run proj tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app
 * \return hipError_t error message(s), if any
 */
template <typename GraphT>
hipError_t RunTests(
    util::Parameters &parameters,
    GraphT           &graph,
    typename GraphT::ValueT *ref_projections,
    util::Location target)
{

    hipError_t retval = hipSuccess;

    typedef typename GraphT::VertexT VertexT;
    typedef typename GraphT::ValueT  ValueT;
    typedef typename GraphT::SizeT   SizeT;
    typedef Problem<GraphT>          ProblemT;
    typedef Enactor<ProblemT>        EnactorT;

    // CLI parameters
    bool quiet_mode = parameters.Get<bool>("quiet");
    bool quick      = parameters.Get<bool>("quick");
    int  num_runs   = parameters.Get<int >("num-runs");
    std::string validation = parameters.Get<std::string>("validation");
    util::Info info("proj", parameters, graph);

    util::CpuTimer cpu_timer, total_timer;
    cpu_timer.Start(); total_timer.Start();

    ValueT *h_projections = new ValueT[graph.nodes * graph.nodes];

    // Allocate problem and enactor on GPU, and initialize them
    ProblemT problem(parameters);
    EnactorT enactor;
    GUARD_CU(problem.Init(graph, target));
    GUARD_CU(enactor.Init(problem, target));

    cpu_timer.Stop();
    parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());

    for (int run_num = 0; run_num < num_runs; ++run_num) {
        GUARD_CU(problem.Reset(
            target
        ));
        GUARD_CU(enactor.Reset(
            target
        ));

        util::PrintMsg("__________________________", !quiet_mode);

        cpu_timer.Start();
        GUARD_CU(enactor.Enact());
        cpu_timer.Stop();
        info.CollectSingleRun(cpu_timer.ElapsedMillis());

        util::PrintMsg("--------------------------\nRun "
            + std::to_string(run_num) + " elapsed: "
            + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = "
            + std::to_string(enactor.enactor_slices[0]
                .enactor_stats.iteration), !quiet_mode);

        if (validation == "each") {
            GUARD_CU(problem.Extract(
                h_projections
            ));
            SizeT num_errors = Validate_Results(
                parameters,
                graph,
                h_projections,
                quick ? NULL : ref_projections,
                false);
        }
    }

    cpu_timer.Start();

    if (validation == "last") {
        GUARD_CU(problem.Extract(
            h_projections
        ));
        SizeT num_errors = Validate_Results(
            parameters,
            graph,
            h_projections,
            quick ? NULL : ref_projections,
            false);
    }

    // compute running statistics
    // TODO: change NULL to problem specific per-vertex visited marker, e.g. h_distances
    // info.ComputeTraversalStats(enactor, (VertexT*)NULL);
    // //Display_Memory_Usage(problem);
    // #ifdef ENABLE_PERFORMANCE_PROFILING
    //     //Display_Performance_Profiling(enactor);
    // #endif

    // Clean up
    GUARD_CU(enactor.Release(target));
    GUARD_CU(problem.Release(target));
    delete[] h_projections; h_projections   = NULL;
    cpu_timer.Stop(); total_timer.Stop();

    info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
    return retval;
}

} // namespace proj
} // namespace app
} // namespace gunrock

// ===========================================================================================
// ========================= CODE BELOW THIS LINE NOT NEEDED FOR TESTS =======================
// ===========================================================================================

// /*
// * @brief Entry of gunrock_template function
// * @tparam     GraphT     Type of the graph
// * @tparam     ValueT     Type of the distances
// * @param[in]  parameters Excution parameters
// * @param[in]  graph      Input graph
// * @param[out] distances  Return shortest distance to source per vertex
// * @param[out] preds      Return predecessors of each vertex
// * \return     double     Return accumulated elapsed times for all runs
// */
// template <typename GraphT, typename ValueT = typename GraphT::ValueT>
// double gunrock_Template(
//     gunrock::util::Parameters &parameters,
//     GraphT &graph
//     // TODO: add problem specific outputs, e.g.:
//     //ValueT **distances
//     )
// {
//     typedef typename GraphT::VertexT VertexT;
//     typedef gunrock::app::Template::Problem<GraphT  > ProblemT;
//     typedef gunrock::app::Template::Enactor<ProblemT> EnactorT;
//     gunrock::util::CpuTimer cpu_timer;
//     gunrock::util::Location target = gunrock::util::DEVICE;
//     double total_time = 0;
//     if (parameters.UseDefault("quiet"))
//         parameters.Set("quiet", true);

//     // Allocate problem and enactor on GPU, and initialize them
//     ProblemT problem(parameters);
//     EnactorT enactor;
//     problem.Init(graph  , target);
//     enactor.Init(problem, target);

//     int num_runs = parameters.Get<int>("num-runs");
//     // TODO: get problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT>>("srcs");
//     // int num_srcs = srcs.size();
//     for (int run_num = 0; run_num < num_runs; ++run_num)
//     {
//         // TODO: problem specific inputs, e.g.:
//         // int src_num = run_num % num_srcs;
//         // VertexT src = srcs[src_num];
//         problem.Reset(/*src,*/ target);
//         enactor.Reset(/*src,*/ target);

//         cpu_timer.Start();
//         enactor.Enact(/*src*/);
//         cpu_timer.Stop();

//         total_time += cpu_timer.ElapsedMillis();
//         // TODO: extract problem specific data, e.g.:
//         problem.Extract(/*distances[src_num]*/);
//     }

//     enactor.Release(target);
//     problem.Release(target);
//     // TODO: problem specific clean ups, e.g.:
//     // srcs.clear();
//     return total_time;
// }


//  * @brief Simple interface take in graph as CSR format
//  * @param[in]  num_nodes   Number of veritces in the input graph
//  * @param[in]  num_edges   Number of edges in the input graph
//  * @param[in]  row_offsets CSR-formatted graph input row offsets
//  * @param[in]  col_indices CSR-formatted graph input column indices
//  * @param[in]  edge_values CSR-formatted graph input edge weights
//  * @param[in]  num_runs    Number of runs to perform SSSP
//  * @param[in]  sources     Sources to begin traverse, one for each run
//  * @param[in]  mark_preds  Whether to output predecessor info
//  * @param[out] distances   Return shortest distance to source per vertex
//  * @param[out] preds       Return predecessors of each vertex
//  * \return     double      Return accumulated elapsed times for all runs

// template <
//     typename VertexT = int,
//     typename SizeT   = int,
//     typename GValueT = unsigned int,
//     typename TValueT = GValueT>
// float Template(
//     const SizeT        num_nodes,
//     const SizeT        num_edges,
//     const SizeT       *row_offsets,
//     const VertexT     *col_indices,
//     const GValueT     *edge_values,
//     const int          num_runs
//     // TODO: add problem specific inputs and outputs, e.g.:
//     //      VertexT     *sources,
//     //      SSSPValueT **distances
//     )
// {
//     // TODO: change to other graph representation, if not using CSR
//     typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
//         gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>
//         GraphT;
//     typedef typename GraphT::CsrT CsrT;

//     // Setup parameters
//     gunrock::util::Parameters parameters("Template");
//     gunrock::graphio::UseParameters(parameters);
//     gunrock::app::Template::UseParameters(parameters);
//     gunrock::app::UseParameters_test(parameters);
//     parameters.Parse_CommandLine(0, NULL);
//     parameters.Set("graph-type", "by-pass");
//     parameters.Set("num-runs", num_runs);
//     // TODO: problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs;
//     // for (int i = 0; i < num_runs; i ++)
//     //     srcs.push_back(sources[i]);
//     // parameters.Set("srcs", srcs);

//     bool quiet = parameters.Get<bool>("quiet");
//     GraphT graph;
//     // Assign pointers into gunrock graph format
//     // TODO: change to other graph representation, if not using CSR
//     graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
//     graph.CsrT::row_offsets   .SetPointer(row_offsets, num_nodes + 1, gunrock::util::HOST);
//     graph.CsrT::column_indices.SetPointer(col_indices, num_edges, gunrock::util::HOST);
//     graph.CsrT::edge_values   .SetPointer(edge_values, num_edges, gunrock::util::HOST);
//     graph.FromCsr(graph.csr(), true, quiet);
//     gunrock::graphio::LoadGraph(parameters, graph);

//     // Run the Template
//     // TODO: add problem specific outputs, e.g.
//     double elapsed_time = gunrock_Template(parameters, graph /*, distances*/);

//     // Cleanup
//     graph.Release();
//     // TODO: problem specific cleanup
//     // srcs.clear();

//     return elapsed_time;
// }

// // Leave this at the end of the file
// // Local Variables:
// // mode:c++
// // c-file-style: "NVIDIA"
// // End:
