// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file sssp_app.cu
 *
 * @brief single-source shortest path (SSSP) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// single-source shortest path includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;

/**
 * @brief SSSP_Parameter structure
 */
struct SSSP_Parameter : gunrock::app::TestParameter_Base
{
public:
    bool   mark_predecessors;
    int    delta_factor;
    double max_queue_sizing1;

    SSSP_Parameter()
    {
        delta_factor      =    32;
        mark_predecessors = false;
        max_queue_sizing1 =  -1.0;
    }

    ~SSSP_Parameter()
    {
    }
};

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 *
 * \return Elapsed run time in milliseconds
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    bool MARK_PREDECESSORS >
float runSSSP(GRGraph* output, SSSP_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 *
 * \return Elapsed run time in milliseconds
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value>
float markPredecessorsSSSP(GRGraph* output, SSSP_Parameter *parameter)
{
    if (parameter->mark_predecessors)
        return runSSSP<VertexId, SizeT, Value, true>(output, parameter);
    else
        return runSSSP<VertexId, SizeT, Value, false>(output, parameter);
}

/**
 * @brief Run test
 *
 * @tparam VertexId          Vertex identifier type*
 * @tparam SizeT             Graph size type
 * @tparam Value             Attribute type
 * @tparam MARK_PREDECESSORS Enable mark predecessors
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 *
 * \return Elapsed run time in milliseconds
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    bool MARK_PREDECESSORS >
float runSSSP(GRGraph* output, SSSP_Parameter *parameter)
{
    typedef SSSPProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS > Problem;

    typedef SSSPEnactor < Problem>
            //INSTRUMENT,
            //DEBUG,
            //SIZE_CHECK >
            Enactor;

    Csr<VertexId, SizeT, Value>
        *graph = (Csr<VertexId, SizeT, Value>*)parameter->graph;
    bool          quiet              = parameter -> g_quiet;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    int           num_iters          = parameter -> iterations;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_queue_sizing1   = parameter -> max_queue_sizing1;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    int           delta_factor       = parameter -> delta_factor;
    std::string   traversal_mode     = parameter -> traversal_mode;
    bool          instrument         = parameter -> instrumented;
    bool          debug              = parameter -> debug;
    bool          size_check         = parameter -> size_check;
    size_t       *org_size           = new size_t[num_gpus];
    // Allocate host-side distance arrays
    Value    *h_distances = new Value[graph->nodes];
    VertexId *h_preds  = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    if (max_queue_sizing < 1.2) max_queue_sizing=1.2;

    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    Problem* problem = new Problem;  // Allocate problem on GPU
    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            delta_factor,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "Problem SSSP Initialization Failed", __FILE__, __LINE__);

    Enactor* enactor = new Enactor(
        num_gpus, gpu_idx, instrument, debug, size_check);  // enactor map
    util::GRError(
        enactor->Init (context, problem, max_grid_size, traversal_mode),
        "SSSP Enactor init failed", __FILE__, __LINE__);

    // Perform SSSP
    CpuTimer cpu_timer;
    float elapsed = 0.0f;
    for (int i = 0; i < num_iters; ++i)
    {
        printf("Round %d of sssp.\n", i+1);

        util::GRError(
                problem->Reset(parameter->src[i], enactor->GetFrontierType(), max_queue_sizing, max_queue_sizing1),
                "SSSP Problem Data Reset Failed", __FILE__, __LINE__);
        util::GRError(
                enactor->Reset(), "SSSP Enactor Reset failed", __FILE__, __LINE__);

        cpu_timer.Start();
        util::GRError(
                enactor->Enact(parameter->src[i], traversal_mode),
                "SSSP Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();

        elapsed += cpu_timer.ElapsedMillis();
    }

    // Copy out results
    util::GRError(
        problem->Extract(h_distances, h_preds),
        "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    output->node_value1 = (Value*)&h_distances[0];
    if (MARK_PREDECESSORS) output->node_value2 = (VertexId*)&h_preds[0];

    if (!quiet)
    {
        printf(" GPU Single-Source Shortest Path finished in %lf msec.\n", elapsed);
    }

    // Clean up
    if (org_size) { delete[] org_size; org_size = NULL; }
    if (enactor ) { delete   enactor ; enactor  = NULL; }
    if (problem ) { delete   problem ; problem  = NULL; }

    return elapsed;
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 *
 * \return Elapsed run time in milliseconds
 */
float dispatchSSSP(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup* config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams)
{
    SSSP_Parameter *parameter = new SSSP_Parameter;
    parameter->iterations = config->num_iters;
    parameter->src = (long long*)malloc(sizeof(long long)*config->num_iters);
    parameter->context  = context;
    parameter->streams  = streams;
    parameter->g_quiet  = config -> quiet;
    parameter->num_gpus = config -> num_devices;
    parameter->gpu_idx  = config -> device_list;
    parameter->delta_factor = config -> delta_factor;
    parameter->traversal_mode = std::string(config -> traversal_mode);
    parameter->mark_predecessors  = config -> mark_predecessors;

    float elapsed_time;

    switch (data_t.VTXID_TYPE)
    {
    case VTXID_INT:
    {
        switch (data_t.SIZET_TYPE)
        {
        case SIZET_INT:
        {
            switch (data_t.VALUE_TYPE)
            {
            case VALUE_INT:    // template type = <int, int, int>
            {
                Csr<int, int, int> csr(false);
                csr.nodes = graphi->num_nodes;
                csr.edges = graphi->num_edges;
                csr.row_offsets    = (int*)graphi->row_offsets;
                csr.column_indices = (int*)graphi->col_indices;
                csr.edge_values    = (int*)graphi->edge_values;
                parameter->graph = &csr;

                // determine source vertex to start
                switch (config -> source_mode)
                {
                case randomize:
                {
                    parameter->src[0] = graphio::RandomNode(csr.nodes);
                    break;
                }
                case largest_degree:
                {
                    int max_deg = 0;
                    parameter->src[0] = csr.GetNodeWithHighestDegree(max_deg);
                    break;
                }
                case manually:
                {
                    parameter->src[0] = config -> source_vertex[0];
                    break;
                }
                default:
                {
                    parameter->src[0] = 0;
                    break;
                }
                }
                if (!parameter->g_quiet)
                {
                    printf(" source: %lld\n", (long long) parameter->src[0]);
                }

                elapsed_time = markPredecessorsSSSP<int, int, int>(grapho, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                csr.edge_values    = NULL;
                break;
            }
            case VALUE_UINT:    // template type = <int, uint, int>
            {
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT:
            {
              Csr<int, int, float> csr(false);
              csr.nodes = graphi->num_nodes;
              csr.edges = graphi->num_edges;
              csr.row_offsets    = (int*)graphi->row_offsets;
              csr.column_indices = (int*)graphi->col_indices;
              csr.edge_values    = (float*)graphi->edge_values;
              parameter->graph = &csr;

              // determine source vertex to start
              switch (config -> source_mode)
              {
              case randomize:
              {
                  parameter->src[0] = graphio::RandomNode(csr.nodes);
                  break;
              }
              case largest_degree:
              {
                  int max_deg = 0;
                  parameter->src[0] = csr.GetNodeWithHighestDegree(max_deg);
                  break;
              }
              case manually:
              {
                  parameter->src[0] = config -> source_vertex[0];
                  break;
              }
              default:
              {
                  parameter->src[0] = 0;
                  break;
              }
              }
              if (!parameter->g_quiet)
              {
                  printf(" source: %lld\n", (long long) parameter->src[0]);
              }

              elapsed_time = markPredecessorsSSSP<int, int, float>(grapho, parameter);

              // reset for free memory
              csr.row_offsets    = NULL;
              csr.column_indices = NULL;
              csr.edge_values    = NULL;

              break;
            }
            }
            break;
        }
        }
        break;
    }
    }
    free(parameter->src);
    return elapsed_time;
}

/*
 * @brief Entry of gunrock_sssp function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
float gunrock_sssp(
    GRGraph*       grapho,
    const GRGraph* graphi,
    const GRSetup* config,
    const GRTypes  data_t)
{
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config -> num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i)
    {
        gpu_idx[i] = config -> device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    if (!config -> quiet) { printf(" using %d GPUs:", num_gpus); }
    for (int gpu = 0; gpu < num_gpus; ++gpu)
    {
        if (!config -> quiet) { printf(" %d ", gpu_idx[gpu]); }
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i)
        {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus)
            {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    if (!config -> quiet) { printf("\n"); }

    return dispatchSSSP(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] distances   Return shortest distance to source per nodes
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  source      Source to begin traverse
 */
float sssp(
    unsigned int*       distances,
    int*                preds,
    const int           num_nodes,
    const int           num_edges,
    const int*          row_offsets,
    const int*          col_indices,
    const unsigned int* edge_values,
    const int           num_iters,
    int*                source,
    const bool          mark_preds)
{
    struct GRTypes data_t;          // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;  // integer vertex identifier
    data_t.SIZET_TYPE = SIZET_INT;  // integer graph size type
    data_t.VALUE_TYPE = VALUE_INT;  // integer attributes type

    struct GRSetup *config = InitSetup(num_iters, source);  // primitive-specific configures
    config -> mark_predecessors = mark_preds;     // do not mark predecessors

    struct GRGraph *grapho = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graphi = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graphi->num_nodes   = num_nodes;  // setting graph nodes
    graphi->num_edges   = num_edges;  // setting graph edges
    graphi->row_offsets = (void*)&row_offsets[0];  // setting row_offsets
    graphi->col_indices = (void*)&col_indices[0];  // setting col_indices
    graphi->edge_values = (void*)&edge_values[0];  // setting edge_values

    float elapsed_time = gunrock_sssp(grapho, graphi, config, data_t);
    memcpy(distances, (int*)grapho->node_value1, num_nodes * sizeof(int));
    if (mark_preds)
        memcpy(preds, (int*)grapho->node_value2, num_nodes * sizeof(int));

    if (graphi) free(graphi);
    if (grapho) free(grapho);
    if (config) free(config);

    return elapsed_time;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
