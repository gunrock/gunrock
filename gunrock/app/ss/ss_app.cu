// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file ss_app.cu
 *
 * @brief scan statistics (SS) application
 */

#include <gunrock/gunrock.h>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph definations
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

// scan statistics includes
#include <gunrock/app/ss/ss_enactor.cuh>
#include <gunrock/app/ss/ss_test.cuh>

namespace gunrock {
namespace app {
namespace ss {

hipError_t UseParameters(util::Parameters &parameters)
{
    hipError_t retval = hipSuccess;
    GUARD_CU(UseParameters_app    (parameters));
    GUARD_CU(UseParameters_problem(parameters));
    GUARD_CU(UseParameters_enactor(parameters));
    GUARD_CU(UseParameters_test   (parameters));

    return retval;
}

/**
 * @brief Run SS tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
 * @param[in]  ref_distances Reference distances
 * @param[in]  target        Whether to perform the SS
 * \return hipError_t error message(s), if any
 */
template <typename GraphT, typename ValueT = typename GraphT::ValueT>
hipError_t RunTests(
    util::Parameters         &parameters,
    GraphT                   &graph,
    typename GraphT::ValueT  *ref_scan_stats,
    util::Location target = util::DEVICE)
{
    hipError_t retval = hipSuccess;
    typedef typename GraphT::VertexT VertexT;
    typedef typename GraphT::SizeT   SizeT;
    typedef Problem<GraphT  > ProblemT;
    typedef Enactor<ProblemT> EnactorT;
    util::CpuTimer    cpu_timer, total_timer;
    cpu_timer.Start(); total_timer.Start();

    // parse configurations from parameters
    bool quiet_mode = parameters.Get<bool>("quiet");
    int  num_runs   = parameters.Get<int >("num-runs");
    std::string validation = parameters.Get<std::string>("validation");
    util::Info info("SS", parameters, graph); // initialize Info structure

    ValueT *h_scan_stats = new ValueT[graph.nodes];

    ProblemT problem(parameters);
    EnactorT enactor;
    GUARD_CU(problem.Init(graph  , target));
    GUARD_CU(enactor.Init(problem, target));
    cpu_timer.Stop();
    parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());

    // perform SS
    for (int run_num = 0; run_num < num_runs; ++run_num)
    {
        GUARD_CU(problem.Reset(target));
        GUARD_CU(enactor.Reset(target));
        util::PrintMsg("__________________________", !quiet_mode);

        cpu_timer.Start();
        GUARD_CU(enactor.Enact());
        cpu_timer.Stop();
        info.CollectSingleRun(cpu_timer.ElapsedMillis());

        util::PrintMsg("--------------------------\nRun "
            + std::to_string(run_num) + " elapsed: "
            + std::to_string(cpu_timer.ElapsedMillis()) + " ms, #iterations = "
            + std::to_string(enactor.enactor_slices[0]
                .enactor_stats.iteration), !quiet_mode);

        if (validation == "each")
        {
            GUARD_CU(problem.Extract(h_scan_stats));
            SizeT num_errors = app::ss::Validate_Results(
                parameters, graph, h_scan_stats,
                ref_scan_stats, false);
        }
    }

    cpu_timer.Start();
    // Copy out results
    GUARD_CU(problem.Extract(h_scan_stats));
    if (validation == "last")
    {
        SizeT num_errors = app::ss::Validate_Results(
            parameters, graph, h_scan_stats,
            ref_scan_stats, false);
    }

    // compute running statistics
    info.ComputeTraversalStats(enactor, (VertexT*)NULL);
    //Display_Memory_Usage(problem);
    #ifdef ENABLE_PERFORMANCE_PROFILING
        //Display_Performance_Profiling(enactor);
    #endif
    // Clean up
    GUARD_CU(enactor.Release(target));
    GUARD_CU(problem.Release(target));
    delete[] h_scan_stats  ; h_scan_stats   = NULL;
    cpu_timer.Stop(); total_timer.Stop();

    info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
    return retval;
}

} // namespace ss
} // namespace app
} // namespace gunrock

/*
 * @brief Entry of gunrock_ss function
 * @tparam     GraphT     Type of the graph
 * @tparam     ValueT     Type of the distances
 * @param[in]  parameters Excution parameters
 * @param[in]  graph      Input graph
 * @param[out] distances  Return shortest distance to source per vertex
 * @param[out] preds      Return predecessors of each vertex
 * \return     double     Return accumulated elapsed times for all runs
 */
template <typename GraphT, typename ValueT = typename GraphT::ValueT>
double gunrock_ss(
    gunrock::util::Parameters &parameters,
    GraphT &graph,
    typename GraphT::VertexT *node,
    ValueT *scan_stat)
{
    typedef typename GraphT::VertexT VertexT;
    typedef gunrock::app::ss::Problem<GraphT  > ProblemT;
    typedef gunrock::app::ss::Enactor<ProblemT> EnactorT;
    gunrock::util::CpuTimer cpu_timer;
    gunrock::util::Location target = gunrock::util::DEVICE;
    double total_time = 0;
    if (parameters.UseDefault("quiet"))
        parameters.Set("quiet", true);

    // Allocate problem and enactor on GPU, and initialize them
    ProblemT problem(parameters);
    EnactorT enactor;
    problem.Init(graph  , target);
    enactor.Init(problem, target);

    int num_runs = parameters.Get<int>("num-runs");
    for (int run_num = 0; run_num < num_runs; ++run_num)
    {
        problem.Reset(target);
        enactor.Reset(target);

        cpu_timer.Start();
        enactor.Enact();
        cpu_timer.Stop();

        total_time += cpu_timer.ElapsedMillis();
        problem.Extract(node, scan_stat, NULL, target);
    }

    enactor.Release(target);
    problem.Release(target);
    return total_time;
}

/*
 * @brief Simple interface take in graph as CSR format
 * @param[in]  num_nodes   Number of veritces in the input graph
 * @param[in]  num_edges   Number of edges in the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  edge_values CSR-formatted graph input edge weights
 * @param[in]  num_runs    Number of runs to perform SS
 * @param[in]  sources     Sources to begin traverse, one for each run
 * @param[in]  mark_preds  Whether to output predecessor info
 * @param[out] distances   Return shortest distance to source per vertex
 * @param[out] preds       Return predecessors of each vertex
 * \return     double      Return accumulated elapsed times for all runs
 */
template <
    typename VertexT = int,
    typename SizeT   = int,
    typename GValueT = unsigned long,
    typename SSValueT = GValueT>
double ss(
    const SizeT        num_nodes,
    const SizeT        num_edges,
    const SizeT       *row_offsets,
    const VertexT     *col_indices,
    const GValueT     *edge_values,
    const int          num_runs,
          VertexT     *nodes,
          SSValueT    *scan_stats)
{
    typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
        gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>
        GraphT;
    typedef typename GraphT::CsrT CsrT;

    // Setup parameters
    gunrock::util::Parameters parameters("ss");
    gunrock::graphio::UseParameters(parameters);
    gunrock::app::ss::UseParameters(parameters);
    gunrock::app::UseParameters_test(parameters);
    parameters.Parse_CommandLine(0, NULL);
    parameters.Set("graph-type", "by-pass");
    parameters.Set("num-runs", num_runs);
    bool quiet = parameters.Get<bool>("quiet");
    GraphT graph;
    // Assign pointers into gunrock graph format
    graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
    graph.CsrT::row_offsets   .SetPointer(row_offsets, num_nodes + 1, gunrock::util::HOST);
    graph.CsrT::column_indices.SetPointer(col_indices, num_edges, gunrock::util::HOST);
    graph.CsrT::edge_values   .SetPointer(edge_values, num_edges, gunrock::util::HOST);
    graph.FromCsr(graph.csr(), true, quiet);
    gunrock::graphio::LoadGraph(parameters, graph);

    // Run the SS
    double elapsed_time = gunrock_ss(parameters, graph, nodes, scan_stats);
    // Cleanup
    graph.Release();

    return elapsed_time;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
