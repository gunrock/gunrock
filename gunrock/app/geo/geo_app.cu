// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file geo_app.cu
 *
 * @brief Geolocation Application
 */

#include <gunrock/gunrock.h>
#include <gunrock/util/test_utils.cuh>

#include <gunrock/graphio/graphio.cuh>
#include <gunrock/graphio/labels.cuh>

#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

#include <gunrock/app/geo/geo_enactor.cuh>
#include <gunrock/app/geo/geo_test.cuh>

namespace gunrock {
namespace app {
namespace geo {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  GUARD_CU(parameters.Use<int>(
      "geo-iter",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      3, "Number of iterations geolocation should run for (default=3).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "spatial-iter",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      1000,
      "Number of maximum iterations spatial median "
      "kernel should run for (default=1000).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "geo-complete",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      false,
      "Run geolocation application until all locations for all nodes are "
      "found, uses an atomic (default=false).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<std::string>(
      "labels-file",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      "", "User locations label file for geolocation app.", __FILE__,
      __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "debug",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      false,
      "Debug label values, this prints out the entire labels array (longitude, "
      "latitude).",
      __FILE__, __LINE__));

  return retval;
}

/**
 * @brief Run geolocation tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app
 * \return hipError_t error message(s), if any
 */
template <typename GraphT, typename ArrayT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     ArrayT &h_latitude, ArrayT &h_longitude,
                     ArrayT &ref_predicted_lat, ArrayT &ref_predicted_lon,
                     util::Location target) {
  hipError_t retval = hipSuccess;

  typedef typename GraphT::VertexT VertexT;
  typedef typename GraphT::ValueT ValueT;
  typedef typename GraphT::SizeT SizeT;
  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;

  // CLI parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");

  int geo_iter = parameters.Get<int>("geo-iter");
  int spatial_iter = parameters.Get<int>("spatial-iter");

  util::PrintMsg("Number of iterations: " + std::to_string(geo_iter),
                 !quiet_mode);

  util::Info info("geolocation", parameters, graph);

  util::CpuTimer cpu_timer, total_timer;
  cpu_timer.Start();
  total_timer.Start();

  // Allocate problem specific host data array to
  // extract device values to host
  ValueT *h_predicted_lat = new ValueT[graph.nodes];
  ValueT *h_predicted_lon = new ValueT[graph.nodes];

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;

  util::PrintMsg("Initializing problem ... ", !quiet_mode);

  GUARD_CU(problem.Init(graph, target));

  util::PrintMsg("Initializing enactor ... ", !quiet_mode);

  GUARD_CU(enactor.Init(problem, target));

  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());

  for (int run_num = 0; run_num < num_runs; ++run_num) {
    GUARD_CU(problem.Reset(h_latitude.GetPointer(util::HOST),
                           h_longitude.GetPointer(util::HOST), geo_iter,
                           spatial_iter, target));
    GUARD_CU(enactor.Reset(target));

    util::PrintMsg("__________________________", !quiet_mode);

    cpu_timer.Start();
    GUARD_CU(enactor.Enact());
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    util::PrintMsg(
        "--------------------------\nRun " + std::to_string(run_num) +
            " elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);

    if (validation == "each") {
      GUARD_CU(problem.Extract(h_predicted_lat, h_predicted_lon));

      SizeT num_errors =
          Validate_Results(parameters, graph, h_predicted_lat, h_predicted_lon,
                           ref_predicted_lat, ref_predicted_lon, false);
    }
  }

  cpu_timer.Start();

  // Extract problem data
  GUARD_CU(problem.Extract(h_predicted_lat, h_predicted_lon));

  if (validation == "last") {
    SizeT num_errors =
        Validate_Results(parameters, graph, h_predicted_lat, h_predicted_lon,
                         ref_predicted_lat, ref_predicted_lon, false);
  }

  // compute running statistics
  info.ComputeTraversalStats(enactor, (VertexT *)NULL);
// Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(enactor);
#endif

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));

  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
  return retval;
}

}  // namespace geo
}  // namespace app
}  // namespace gunrock

// ===========================================================================================
// ========================= CODE BELOW THIS LINE NOT NEEDED FOR TESTS
// =======================
// ===========================================================================================

// /*
// * @brief Entry of gunrock_template function
// * @tparam     GraphT     Type of the graph
// * @tparam     ValueT     Type of the distances
// * @param[in]  parameters Excution parameters
// * @param[in]  graph      Input graph
// * @param[out] distances  Return shortest distance to source per vertex
// * @param[out] preds      Return predecessors of each vertex
// * \return     double     Return accumulated elapsed times for all runs
// */
// template <typename GraphT, typename ValueT = typename GraphT::ValueT>
// double gunrock_Template(
//     gunrock::util::Parameters &parameters,
//     GraphT &graph
//     // TODO: add problem specific outputs, e.g.:
//     //ValueT **distances
//     )
// {
//     typedef typename GraphT::VertexT VertexT;
//     typedef gunrock::app::Template::Problem<GraphT  > ProblemT;
//     typedef gunrock::app::Template::Enactor<ProblemT> EnactorT;
//     gunrock::util::CpuTimer cpu_timer;
//     gunrock::util::Location target = gunrock::util::DEVICE;
//     double total_time = 0;
//     if (parameters.UseDefault("quiet"))
//         parameters.Set("quiet", true);

//     // Allocate problem and enactor on GPU, and initialize them
//     ProblemT problem(parameters);
//     EnactorT enactor;
//     problem.Init(graph  , target);
//     enactor.Init(problem, target);

//     int num_runs = parameters.Get<int>("num-runs");
//     // TODO: get problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs =
//     parameters.Get<std::vector<VertexT>>("srcs");
//     // int num_srcs = srcs.size();
//     for (int run_num = 0; run_num < num_runs; ++run_num)
//     {
//         // TODO: problem specific inputs, e.g.:
//         // int src_num = run_num % num_srcs;
//         // VertexT src = srcs[src_num];
//         problem.Reset(/*src,*/ target);
//         enactor.Reset(/*src,*/ target);

//         cpu_timer.Start();
//         enactor.Enact(/*src*/);
//         cpu_timer.Stop();

//         total_time += cpu_timer.ElapsedMillis();
//         // TODO: extract problem specific data, e.g.:
//         problem.Extract(/*distances[src_num]*/);
//     }

//     enactor.Release(target);
//     problem.Release(target);
//     // TODO: problem specific clean ups, e.g.:
//     // srcs.clear();
//     return total_time;
// }

//  * @brief Simple interface take in graph as CSR format
//  * @param[in]  num_nodes   Number of veritces in the input graph
//  * @param[in]  num_edges   Number of edges in the input graph
//  * @param[in]  row_offsets CSR-formatted graph input row offsets
//  * @param[in]  col_indices CSR-formatted graph input column indices
//  * @param[in]  edge_values CSR-formatted graph input edge weights
//  * @param[in]  num_runs    Number of runs to perform SSSP
//  * @param[in]  sources     Sources to begin traverse, one for each run
//  * @param[in]  mark_preds  Whether to output predecessor info
//  * @param[out] distances   Return shortest distance to source per vertex
//  * @param[out] preds       Return predecessors of each vertex
//  * \return     double      Return accumulated elapsed times for all runs

// template <
//     typename VertexT = int,
//     typename SizeT   = int,
//     typename GValueT = unsigned int,
//     typename TValueT = GValueT>
// float Geolocation(
//     const SizeT        num_nodes,
//     const SizeT        num_edges,
//     const SizeT       *row_offsets,
//     const VertexT     *col_indices,
//     const GValueT     *edge_values,
//     const int          num_runs
//     // TODO: add problem specific inputs and outputs, e.g.:
//     //      VertexT     *sources,
//     //      SSSPValueT **distances
//     )
// {
//     // TODO: change to other graph representation, if not using CSR
//     typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
//         gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>
//         GraphT;
//     typedef typename GraphT::CsrT CsrT;

//     // Setup parameters
//     gunrock::util::Parameters parameters("Template");
//     gunrock::graphio::UseParameters(parameters);
//     gunrock::app::Template::UseParameters(parameters);
//     gunrock::app::UseParameters_test(parameters);
//     parameters.Parse_CommandLine(0, NULL);
//     parameters.Set("graph-type", "by-pass");
//     parameters.Set("num-runs", num_runs);
//     // TODO: problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs;
//     // for (int i = 0; i < num_runs; i ++)
//     //     srcs.push_back(sources[i]);
//     // parameters.Set("srcs", srcs);

//     bool quiet = parameters.Get<bool>("quiet");
//     GraphT graph;
//     // Assign pointers into gunrock graph format
//     // TODO: change to other graph representation, if not using CSR
//     graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
//     graph.CsrT::row_offsets   .SetPointer(row_offsets, num_nodes + 1,
//     gunrock::util::HOST); graph.CsrT::column_indices.SetPointer(col_indices,
//     num_edges, gunrock::util::HOST); graph.FromCsr(graph.csr(), true, quiet);
//     gunrock::graphio::LoadGraph(parameters, graph);

//     // Run the Template
//     // TODO: add problem specific outputs, e.g.
//     double elapsed_time = gunrock_Template(parameters, graph /*,
//     distances*/);

//     // Cleanup
//     graph.Release();
//     // TODO: problem specific cleanup
//     // srcs.clear();

//     return elapsed_time;
// }

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// // End:
