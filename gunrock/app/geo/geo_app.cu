// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file hello_app.cu
 *
 * @brief Simple Gunrock Application
 */

#include <gunrock/gunrock.h>
#include <gunrock/util/test_utils.cuh>

#include <gunrock/graphio/graphio.cuh>
#include <gunrock/graphio/labels.cuh>

#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

// <DONE> change includes
#include <gunrock/app/geo/geo_enactor.cuh>
#include <gunrock/app/geo/geo_test.cuh>
// </DONE>

namespace gunrock {
namespace app {
// <DONE> change namespace
namespace geo {
// </DONE>


hipError_t UseParameters(util::Parameters &parameters)
{
    hipError_t retval = hipSuccess;
    GUARD_CU(UseParameters_app(parameters));
    GUARD_CU(UseParameters_problem(parameters));
    GUARD_CU(UseParameters_enactor(parameters));

    // <TODO> add app specific parameters, eg:
    // GUARD_CU(parameters.Use<std::string>(
    //    "src",
    //    util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER,
    //    "0",
    //    "<Vertex-ID|random|largestdegree> The source vertices\n"
    //    "\tIf random, randomly select non-zero degree vertices;\n"
    //    "\tIf largestdegree, select vertices with largest degrees",
    //    __FILE__, __LINE__));
    // </TODO>

    GUARD_CU(parameters.Use<std::string>(
        "labels-file",
        util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
        "",
        " labels file.",
        __FILE__, __LINE__));

    return retval;
}

/**
 * @brief Run geolocation tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app 
 * \return hipError_t error message(s), if any
 */
template <typename GraphT>
hipError_t RunTests(
    util::Parameters &parameters,
    GraphT           &graph,
    typename GraphT::ValueT *h_latitude,
    typename GraphT::ValueT *h_longitude,
    // <DONE> add problem specific reference results, e.g.:
    typename GraphT::ValueT *ref_predicted_lat,
    typename GraphT::ValueT *ref_predicted_lon,
    // </DONE>
    util::Location target)
{
    
    hipError_t retval = hipSuccess;
       
    typedef typename GraphT::VertexT VertexT;
    typedef typename GraphT::ValueT  ValueT;
    typedef typename GraphT::SizeT   SizeT;
    typedef Problem<GraphT>          ProblemT;
    typedef Enactor<ProblemT>        EnactorT;

    // CLI parameters
    bool quiet_mode 		= parameters.Get<bool>("quiet");
    int  num_runs   		= parameters.Get<int >("num-runs");
    std::string validation 	= parameters.Get<std::string>("validation");

    util::Info info("geolocation", parameters, graph);

/*
    util::PrintMsg("Labels File Input: "
            + labels_file, !quiet_mode);   

 
    ValueT *h_latitude  = new ValueT[graph.nodes];
    ValueT *h_longitude = new ValueT[graph.nodes];

    retval = gunrock::graphio::labels::Read(parameters, h_latitude, h_longitude);


    util::PrintMsg("Debugging Labels -------------", !quiet_mode);
    for (int p = 0; p < graph.nodes; p++) 
    {
    	util::PrintMsg("    locations[ " + std::to_string(p) + 
			    " ] = < " + std::to_string(h_latitude[p]) +
			    " , " + std::to_string(h_longitude[p]) +
			    " > ",
			    !quiet_mode);
    }
*/
    util::CpuTimer cpu_timer, total_timer;
    cpu_timer.Start(); total_timer.Start();

    // <TODO> get problem specific inputs, e.g.:
    // std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT>>("srcs");
    // printf("RunTests: %d srcs: src[0]=%d\n", srcs.size(), srcs[0]);
    // </TODO>

    // <DONE> allocate problem specific host data, e.g.:
    ValueT *h_predicted_lat = new ValueT[graph.nodes];
    ValueT *h_predicted_lon = new ValueT[graph.nodes];
    // </DONE>

    // Allocate problem and enactor on GPU, and initialize them
    ProblemT problem(parameters);
    EnactorT enactor;

    util::PrintMsg("Initializing problem ... ", !quiet_mode);

    GUARD_CU(problem.Init(graph, target));

    util::PrintMsg("Initializing enactor ... ", !quiet_mode);

    GUARD_CU(enactor.Init(problem, target));
    
    cpu_timer.Stop();
    parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());
    
    for (int run_num = 0; run_num < num_runs; ++run_num) {
        GUARD_CU(problem.Reset(
            // <DONE> problem specific data if necessary, eg:
            h_latitude,
	    h_longitude,
            // </DONE>
            target
        ));
        GUARD_CU(enactor.Reset(
            // <TODO> problem specific data if necessary:
            // srcs[run_num % srcs.size()],
            // </TODO>
            target
        ));
        
        util::PrintMsg("__________________________", !quiet_mode);

        cpu_timer.Start();
        GUARD_CU(enactor.Enact(
            // <TODO> problem specific data if necessary:
            // srcs[run_num % srcs.size()]
            // </TODO>
        ));
        cpu_timer.Stop();
        info.CollectSingleRun(cpu_timer.ElapsedMillis());

        util::PrintMsg("--------------------------\nRun "
            + std::to_string(run_num) + " elapsed: "
            + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = "
            + std::to_string(enactor.enactor_slices[0]
                .enactor_stats.iteration), !quiet_mode);
        
        if (validation == "each") {
            
            GUARD_CU(problem.Extract(
                // <DONE> problem specific data
                h_predicted_lat,
		h_predicted_lon
                // </DONE>
            ));
            SizeT num_errors = Validate_Results(
                parameters,
                graph,
                // <DONE> problem specific data
                h_predicted_lat, h_predicted_lon,
		ref_predicted_lat, ref_predicted_lon,
                // </DONE>
                false);
        }
    }

    cpu_timer.Start();
    
    GUARD_CU(problem.Extract(
        // <DONE> problem specific data
        h_predicted_lat,
	h_predicted_lon
        // </DONE>
    ));
    if (validation == "last") {
        SizeT num_errors = Validate_Results(
            parameters,
            graph,
            // <DONE> problem specific data
            h_predicted_lat, h_predicted_lon,
	    ref_predicted_lat, ref_predicted_lon,
            // </DONE>
            false);
    }

    // compute running statistics
    // TODO: change NULL to problem specific per-vertex visited marker, e.g. h_distances
    info.ComputeTraversalStats(enactor, (VertexT*)NULL);
    //Display_Memory_Usage(problem);
    #ifdef ENABLE_PERFORMANCE_PROFILING
        //Display_Performance_Profiling(enactor);
    #endif

    // Clean up
    GUARD_CU(enactor.Release(target));
    GUARD_CU(problem.Release(target));
    // <DONE> Release problem specific data, e.g.:
    delete[] h_predicted_lat; h_predicted_lat   = NULL;
    delete[] h_predicted_lon; h_predicted_lon   = NULL;
    delete[] h_latitude; h_latitude   = NULL;
    delete[] h_longitude; h_longitude = NULL;
    // </DONE>
    cpu_timer.Stop(); total_timer.Stop();

    info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
    return retval;
}

} // namespace geo
} // namespace app
} // namespace gunrock

// ===========================================================================================
// ========================= CODE BELOW THIS LINE NOT NEEDED FOR TESTS =======================
// ===========================================================================================

// /*
// * @brief Entry of gunrock_template function
// * @tparam     GraphT     Type of the graph
// * @tparam     ValueT     Type of the distances
// * @param[in]  parameters Excution parameters
// * @param[in]  graph      Input graph
// * @param[out] distances  Return shortest distance to source per vertex
// * @param[out] preds      Return predecessors of each vertex
// * \return     double     Return accumulated elapsed times for all runs
// */
// template <typename GraphT, typename ValueT = typename GraphT::ValueT>
// double gunrock_Template(
//     gunrock::util::Parameters &parameters,
//     GraphT &graph
//     // TODO: add problem specific outputs, e.g.:
//     //ValueT **distances
//     )
// {
//     typedef typename GraphT::VertexT VertexT;
//     typedef gunrock::app::Template::Problem<GraphT  > ProblemT;
//     typedef gunrock::app::Template::Enactor<ProblemT> EnactorT;
//     gunrock::util::CpuTimer cpu_timer;
//     gunrock::util::Location target = gunrock::util::DEVICE;
//     double total_time = 0;
//     if (parameters.UseDefault("quiet"))
//         parameters.Set("quiet", true);

//     // Allocate problem and enactor on GPU, and initialize them
//     ProblemT problem(parameters);
//     EnactorT enactor;
//     problem.Init(graph  , target);
//     enactor.Init(problem, target);

//     int num_runs = parameters.Get<int>("num-runs");
//     // TODO: get problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT>>("srcs");
//     // int num_srcs = srcs.size();
//     for (int run_num = 0; run_num < num_runs; ++run_num)
//     {
//         // TODO: problem specific inputs, e.g.:
//         // int src_num = run_num % num_srcs;
//         // VertexT src = srcs[src_num];
//         problem.Reset(/*src,*/ target);
//         enactor.Reset(/*src,*/ target);

//         cpu_timer.Start();
//         enactor.Enact(/*src*/);
//         cpu_timer.Stop();

//         total_time += cpu_timer.ElapsedMillis();
//         // TODO: extract problem specific data, e.g.:
//         problem.Extract(/*distances[src_num]*/);
//     }

//     enactor.Release(target);
//     problem.Release(target);
//     // TODO: problem specific clean ups, e.g.:
//     // srcs.clear();
//     return total_time;
// }


//  * @brief Simple interface take in graph as CSR format
//  * @param[in]  num_nodes   Number of veritces in the input graph
//  * @param[in]  num_edges   Number of edges in the input graph
//  * @param[in]  row_offsets CSR-formatted graph input row offsets
//  * @param[in]  col_indices CSR-formatted graph input column indices
//  * @param[in]  edge_values CSR-formatted graph input edge weights
//  * @param[in]  num_runs    Number of runs to perform SSSP
//  * @param[in]  sources     Sources to begin traverse, one for each run
//  * @param[in]  mark_preds  Whether to output predecessor info
//  * @param[out] distances   Return shortest distance to source per vertex
//  * @param[out] preds       Return predecessors of each vertex
//  * \return     double      Return accumulated elapsed times for all runs
 
// template <
//     typename VertexT = int,
//     typename SizeT   = int,
//     typename GValueT = unsigned int,
//     typename TValueT = GValueT>
// float Template(
//     const SizeT        num_nodes,
//     const SizeT        num_edges,
//     const SizeT       *row_offsets,
//     const VertexT     *col_indices,
//     const GValueT     *edge_values,
//     const int          num_runs
//     // TODO: add problem specific inputs and outputs, e.g.:
//     //      VertexT     *sources,
//     //      SSSPValueT **distances
//     )
// {
//     // TODO: change to other graph representation, if not using CSR
//     typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
//         gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>
//         GraphT;
//     typedef typename GraphT::CsrT CsrT;

//     // Setup parameters
//     gunrock::util::Parameters parameters("Template");
//     gunrock::graphio::UseParameters(parameters);
//     gunrock::app::Template::UseParameters(parameters);
//     gunrock::app::UseParameters_test(parameters);
//     parameters.Parse_CommandLine(0, NULL);
//     parameters.Set("graph-type", "by-pass");
//     parameters.Set("num-runs", num_runs);
//     // TODO: problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs;
//     // for (int i = 0; i < num_runs; i ++)
//     //     srcs.push_back(sources[i]);
//     // parameters.Set("srcs", srcs);

//     bool quiet = parameters.Get<bool>("quiet");
//     GraphT graph;
//     // Assign pointers into gunrock graph format
//     // TODO: change to other graph representation, if not using CSR
//     graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
//     graph.CsrT::row_offsets   .SetPointer(row_offsets, gunrock::util::HOST);
//     graph.CsrT::column_indices.SetPointer(col_indices, gunrock::util::HOST);
//     graph.CsrT::edge_values   .SetPointer(edge_values, gunrock::util::HOST);
//     graph.FromCsr(graph.csr(), true, quiet);
//     gunrock::graphio::LoadGraph(parameters, graph);

//     // Run the Template
//     // TODO: add problem specific outputs, e.g.
//     double elapsed_time = gunrock_Template(parameters, graph /*, distances*/);

//     // Cleanup
//     graph.Release();
//     // TODO: problem specific cleanup
//     // srcs.clear();

//     return elapsed_time;
// }

// // Leave this at the end of the file
// // Local Variables:
// // mode:c++
// // c-file-style: "NVIDIA"
// // End:
