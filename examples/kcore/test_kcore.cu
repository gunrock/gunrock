// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_kcore.cu
 *
 * @brief K-Core driver program for Gunrock.
 */

#include <gunrock/app/kcore/kcore_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    typedef typename app::TestGraph<VertexT, SizeT, ValueT,
                                    graph::HAS_CSR>
        GraphT;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    GraphT graph;  // graph we process on

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, graph));
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // reference result on CPU:
    VertexT *ref_num_cores = NULL;

    bool quick = parameters.Get<bool>("quick");

    // compute reference CPU K-Core
    if (!quick) {
      bool quiet = parameters.Get<bool>("quiet");
      std::string validation = parameters.Get<std::string>("validation");
      util::PrintMsg("Computing reference value ...", !quiet);

      // problem specific data:
      ref_num_cores = new VertexT[graph.nodes];

      util::PrintMsg("__________________________", !quiet);
      float elapsed =
          app::kcore::CPU_Reference(parameters, graph.csr(), ref_num_cores);

      util::PrintMsg(
          "--------------------------\n Elapsed: " + std::to_string(elapsed),
          !quiet);
    }

    std::vector<std::string> switches{"advance-mode"};
    GUARD_CU(app::Switch_Parameters(
        parameters, graph, switches,
        [ref_num_cores](util::Parameters &parameters,
                                    GraphT &graph) {
          return app::kcore::RunTests(parameters, graph,
                                      ref_num_cores, util::DEVICE);
        }));

    if (!quick) {
      delete[] ref_num_cores;
      ref_num_cores = NULL;
    }
    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test kcore");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::kcore::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B | app::SIZET_U32B |
                           app::VALUET_F32B | app::UNDIRECTED>(parameters,
                                                               main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End: