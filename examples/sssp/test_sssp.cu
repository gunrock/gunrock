#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for single source shortest path.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// SSSP includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>


#include <gunrock/app/sample/sample_enactor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <gunrock/priority_queue/kernel.cuh>

#include <gunrock/util/shared_utils.cuh>

#include <moderngpu.cuh>

// Boost includes for CPU Dijkstra SSSP reference algorithms
#include <boost/config.hpp>
#include <boost/graph/graph_traits.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/property_map/property_map.hpp>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        "test <graph-type> [graph-type-arguments]\n"
        "Graph type and graph type arguments:\n"
        "    market <matrix-market-file-name>\n"
        "        Reads a Matrix-Market coordinate-formatted graph of\n"
        "        directed/undirected edges from STDIN (or from the\n"
        "        optionally-specified file).\n"
        "    rmat (default: rmat_scale = 10, a = 0.57, b = c = 0.19)\n"
        "        Generate R-MAT graph as input\n"
        "        --rmat_scale=<vertex-scale>\n"
        "        --rmat_nodes=<number-nodes>\n"
        "        --rmat_edgefactor=<edge-factor>\n"
        "        --rmat_edges=<number-edges>\n"
        "        --rmat_a=<factor> --rmat_b=<factor> --rmat_c=<factor>\n"
        "        --rmat_seed=<seed>\n"
        "    rgg (default: rgg_scale = 10, rgg_thfactor = 0.55)\n"
        "        Generate Random Geometry Graph as input\n"
        "        --rgg_scale=<vertex-scale>\n"
        "        --rgg_nodes=<number-nodes>\n"
        "        --rgg_thfactor=<threshold-factor>\n"
        "        --rgg_threshold=<threshold>\n"
        "        --rgg_vmultipiler=<vmultipiler>\n"
        "        --rgg_seed=<seed>\n\n"
        "Optional arguments:\n"
        "[--device=<device_index>] Set GPU(s) for testing (Default: 0).\n"
        "[--undirected]            Treat the graph as undirected (symmetric).\n"
        "[--instrumented]          Keep kernels statics [Default: Disable].\n"
        "                          total_queued, search_depth and barrier duty.\n"
        "                          (a relative indicator of load imbalance.)\n"
        "[--src=<Vertex-ID|randomize|largestdegree>]\n"
        "                          Begins traversal from the source (Default: 0).\n"
        "                          If randomize: from a random source vertex.\n"
        "                          If largestdegree: from largest degree vertex.\n"
        "[--quick]                 Skip the CPU reference validation process.\n"
        "[--mark-pred]             Keep both label info and predecessor info.\n"
        "[--disable-size-check]    Disable frontier queue size check.\n"
        "[--grid-size=<grid size>] Maximum allowed grid size setting.\n"
        "[--queue-sizing=<factor>] Allocates a frontier queue sized at: \n"
        "                          (graph-edges * <factor>). (Default: 1.0)\n"
        "[--in-sizing=<in/out_queue_scale_factor>]\n"
        "                          Allocates a frontier queue sized at: \n"
        "                          (graph-edges * <factor>). (Default: 1.0)\n"
        "[--v]                     Print verbose per iteration debug info.\n"
        "[--iteration-num=<num>]   Number of runs to perform the test.\n"
        "[--traversal-mode=<0|1>]  Set traversal strategy, 0 for Load-Balanced\n"
        "                          1 for Dynamic-Cooperative (Default: dynamic\n"
        "                          determine based on average degree).\n"
        "[--partition-method=<random|biasrandom|clustered|metis>]\n"
        "                          Choose partitioner (Default use random).\n"
        "[--delta_factor=<factor>] Delta factor for delta-stepping SSSP.\n"
        "[--quiet]                 No output (unless --json is specified).\n"
        "[--json]                  Output JSON-format statistics to STDOUT.\n"
        "[--jsonfile=<name>]       Output JSON-format statistics to file <name>\n"
        "[--jsondir=<dir>]         Output JSON-format statistics to <dir>/name,\n"
        "                          where name is auto-generated.\n"
    );
}

/**
 * @brief Displays the SSSP result (i.e., distance from source)
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] num_nodes Number of nodes in the graph.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution (VertexId *source_path, SizeT num_nodes)
{
    if (num_nodes > 40) num_nodes = 40;

    printf("[");
    for (VertexId i = 0; i < num_nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(" ");
    }
    printf("]\n");
}

/******************************************************************************
 * SSSP Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference SSSP ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] node_values Host-side vector to store CPU computed labels for each node
 * @param[in] node_preds Host-side vector to store CPU computed predecessors for each node
 * @param[in] src Source node where SSSP starts
 * @param[in] quiet Don't print out anything to stdout
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    bool     MARK_PREDECESSORS >
void ReferenceSssp(
    const Csr<VertexId, SizeT, Value> &graph,
    Value                             *node_values,
    VertexId                          *node_preds,
    VertexId                          src,
    bool                              quiet)
{
    using namespace boost;

    // Prepare Boost Datatype and Data structure
    typedef adjacency_list<vecS, vecS, directedS, no_property,
            property <edge_weight_t, unsigned int> > Graph;

    typedef graph_traits<Graph>::vertex_descriptor vertex_descriptor;
    typedef graph_traits<Graph>::edge_descriptor edge_descriptor;

    typedef std::pair<VertexId, VertexId> Edge;

    Edge   *edges = ( Edge*)malloc(sizeof( Edge) * graph.edges);
    Value *weight = (Value*)malloc(sizeof(Value) * graph.edges);

    for (SizeT i = 0; i < graph.nodes; ++i)
    {
        for (SizeT j = graph.row_offsets[i]; j < graph.row_offsets[i + 1]; ++j)
        {
            edges[j] = Edge(i, graph.column_indices[j]);
            weight[j] = graph.edge_values[j];
        }
    }

    Graph g(edges, edges + graph.edges, weight, graph.nodes);

    std::vector<Value> d(graph.nodes);
    std::vector<vertex_descriptor> p(graph.nodes);
    vertex_descriptor s = vertex(src, g);

    property_map<Graph, vertex_index_t>::type indexmap = get(vertex_index, g);

    //
    // Perform SSSP
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    if (MARK_PREDECESSORS)
    {
        dijkstra_shortest_paths(g, s,
            predecessor_map(boost::make_iterator_property_map(
                p.begin(), get(boost::vertex_index, g))).distance_map(
                    boost::make_iterator_property_map(
                        d.begin(), get(boost::vertex_index, g))));
    }
    else
    {
        dijkstra_shortest_paths(g, s,
            distance_map(boost::make_iterator_property_map(
                d.begin(), get(boost::vertex_index, g))));
    }
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    if (!quiet) { printf("CPU SSSP finished in %lf msec.\n", elapsed); }

    Coo<Value, Value>* sort_dist = NULL;
    Coo<VertexId, VertexId>* sort_pred = NULL;
    sort_dist = (Coo<Value, Value>*)malloc(
                    sizeof(Coo<Value, Value>) * graph.nodes);
    if (MARK_PREDECESSORS)
    {
        sort_pred = (Coo<VertexId, VertexId>*)malloc(
                        sizeof(Coo<VertexId, VertexId>) * graph.nodes);
    }
    graph_traits < Graph >::vertex_iterator vi, vend;
    for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
    {
        sort_dist[(*vi)].row = (*vi);
        sort_dist[(*vi)].col = d[(*vi)];
    }
    std::stable_sort(
        sort_dist, sort_dist + graph.nodes,
        RowFirstTupleCompare<Coo<Value, Value> >);

    if (MARK_PREDECESSORS)
    {
        for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
        {
            sort_pred[(*vi)].row = (*vi);
            sort_pred[(*vi)].col = p[(*vi)];
        }
        std::stable_sort(
            sort_pred, sort_pred + graph.nodes,
            RowFirstTupleCompare< Coo<VertexId, VertexId> >);
    }

    for (SizeT i = 0; i < graph.nodes; ++i)
    {
        node_values[i] = sort_dist[i].col;
    }
    if (MARK_PREDECESSORS)
    {
        for (SizeT i = 0; i < graph.nodes; ++i)
        {
            node_preds[i] = sort_pred[i].col;
        }
    }
    if (sort_dist) free(sort_dist);
    if (sort_pred) free(sort_pred);
}


/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] info Pointer to info contains parameters and statistics.
 *
 * \return hipError_t object which indicates the success of
 * all CUDA function calls.
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    bool MARK_PREDECESSORS >
hipError_t RunTests(Info<VertexId, SizeT, Value> *info)
{
    typedef SSSPProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS > Problem;

    typedef SSSPEnactor < Problem > Enactor;

    // parse configurations from mObject info
    Csr<VertexId, SizeT, Value> *graph = info->csr_ptr;
    VertexId    src                 = info->info["source_vertex"    ].get_int64();
    int         max_grid_size       = info->info["max_grid_size"    ].get_int  ();
    int         num_gpus            = info->info["num_gpus"         ].get_int  ();
    double      max_queue_sizing    = info->info["max_queue_sizing" ].get_real ();
    double      max_queue_sizing1   = info->info["max_queue_sizing1"].get_real ();
    double      max_in_sizing       = info->info["max_in_sizing"    ].get_real ();
    std::string partition_method    = info->info["partition_method" ].get_str  ();
    double      partition_factor    = info->info["partition_factor" ].get_real ();
    int         partition_seed      = info->info["partition_seed"   ].get_int  ();
    bool        quiet_mode          = info->info["quiet_mode"       ].get_bool ();
    bool        quick_mode          = info->info["quick_mode"       ].get_bool ();
    bool        stream_from_host    = info->info["stream_from_host" ].get_bool ();
    std::string traversal_mode      = info->info["traversal_mode"   ].get_str  ();
    bool        instrument          = info->info["instrument"       ].get_bool ();
    bool        debug               = info->info["debug_mode"       ].get_bool ();
    bool        size_check          = info->info["size_check"       ].get_bool ();
    int         iterations          = info->info["num_iteration"    ].get_int  ();
    int         delta_factor        = info->info["delta_factor"     ].get_int  ();
    std::string src_type            = info->info["source_type"      ].get_str  ();
    int      src_seed               = info->info["source_seed"      ].get_int  ();
    int      communicate_latency    = info->info["communicate_latency"].get_int ();
    float    communicate_multipy    = info->info["communicate_multipy"].get_real();
    int      expand_latency         = info->info["expand_latency"    ].get_int ();
    int      subqueue_latency       = info->info["subqueue_latency"  ].get_int ();
    int      fullqueue_latency      = info->info["fullqueue_latency" ].get_int ();
    int      makeout_latency        = info->info["makeout_latency"   ].get_int ();
    if (max_queue_sizing < 1.2) max_queue_sizing=1.2;
    if (max_in_sizing < 0) max_in_sizing = 1.0;
    if (communicate_multipy > 1) max_in_sizing *= communicate_multipy;

    CpuTimer    cpu_timer;
    hipError_t retval              = hipSuccess;

    cpu_timer.Start();
    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr   *context = (ContextPtr*)  info->context;
    hipStream_t *streams = (hipStream_t*)info->streams;

    // Allocate host-side array (for both reference and GPU-computed results)
    Value    *reference_labels      = new Value[graph->nodes];
    Value    *h_labels              = new Value[graph->nodes];
    Value    *reference_check_label = (quick_mode) ? NULL : reference_labels;
    VertexId *reference_preds       = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId *h_preds               = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId *reference_check_pred  = (quick_mode || !MARK_PREDECESSORS) ? NULL : reference_preds;

    size_t *org_size = new size_t[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        if (retval = util::SetDevice(gpu_idx[gpu])) return retval;
        if (retval = util::GRError(hipMemGetInfo(&(org_size[gpu]), &dummy),
            "hipMemGetInfo failed", __FILE__, __LINE__)) return retval;
    }

    // Allocate problem on GPU
    Problem *problem = new Problem;
    if (retval = util::GRError(problem->Init(
        stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        delta_factor,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed),
        "SSSP Problem Init failed", __FILE__, __LINE__))
        return retval;

    // Allocate SSSP enactor map
    Enactor* enactor = new Enactor(
        num_gpus, gpu_idx, instrument, debug, size_check);
    if (retval = util::GRError(enactor->Init(
        context, problem, max_grid_size, traversal_mode),
        "SSSP Enactor Init failed", __FILE__, __LINE__))
        return retval;

    enactor -> communicate_latency = communicate_latency;
    enactor -> communicate_multipy = communicate_multipy;
    enactor -> expand_latency      = expand_latency;
    enactor -> subqueue_latency    = subqueue_latency;
    enactor -> fullqueue_latency   = fullqueue_latency;
    enactor -> makeout_latency     = makeout_latency;

    if (retval = util::SetDevice(gpu_idx[0])) return retval;
    if (retval = util::latency::Test(
        streams[0], problem -> data_slices[0] -> latency_data,
        communicate_latency,
        communicate_multipy,
        expand_latency,
        subqueue_latency,
        fullqueue_latency,
        makeout_latency)) return retval;

    cpu_timer.Stop();
    info -> info["preprocess_time"] = cpu_timer.ElapsedMillis();

    // perform SSSP
    double total_elapsed  = 0.0;
    double single_elapsed = 0.0;
    double max_elapsed    = 0.0;
    double min_elapsed    = 1e10;
    json_spirit::mArray process_times;
    if (src_type == "random2")
    {
        if (src_seed == -1) src_seed = time(NULL);
        if (!quiet_mode)
            printf("src_seed = %d\n", src_seed);
        srand(src_seed);
    }
    if (!quiet_mode) printf("Using traversal mode %s\n", traversal_mode.c_str());
    for (int iter = 0; iter < iterations; ++iter)
    {
        if (src_type == "random2")
        {
            bool src_valid = false;
            while (!src_valid)
            {
                src = rand() % graph -> nodes;
                if (graph -> row_offsets[src] != graph -> row_offsets[src+1])
                    src_valid = true;
            }
        }

        if (retval = util::GRError(problem->Reset(
            src, enactor->GetFrontierType(),
            max_queue_sizing, max_queue_sizing1),
            "SSSP Problem Data Reset Failed", __FILE__, __LINE__))
            return retval;

        if (retval = util::GRError(enactor->Reset(),
            "SSSP Enactor Reset failed", __FILE__, __LINE__))
            return retval;

        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            if (retval = util::SetDevice(gpu_idx[gpu]))
                return retval;
            if (retval = util::GRError(hipDeviceSynchronize(),
                "hipDeviceSynchronize failed", __FILE__, __LINE__))
                return retval;
        }

        if (!quiet_mode)
        {
            printf("__________________________\n"); fflush(stdout);
        }
        cpu_timer.Start();
        if (retval = util::GRError(enactor->Enact(src, traversal_mode),
            "SSSP Problem Enact Failed", __FILE__, __LINE__))
            return retval;
        cpu_timer.Stop();
        single_elapsed = cpu_timer.ElapsedMillis();
        total_elapsed += single_elapsed;
        process_times.push_back(single_elapsed);
        if (single_elapsed > max_elapsed) max_elapsed = single_elapsed;
        if (single_elapsed < min_elapsed) min_elapsed = single_elapsed;
        if (!quiet_mode)
        {
            printf("--------------------------\n"
                "iteration %d elapsed: %lf ms, src = %lld, #iteration = %lld\n",
                iter, single_elapsed, (long long)src,
                (long long)enactor -> enactor_stats -> iteration);
            fflush(stdout);
        }
    }
    total_elapsed /= iterations;
    info -> info["process_times"] = process_times;
    info -> info["min_process_time"] = min_elapsed;
    info -> info["max_process_time"] = max_elapsed;

    // compute reference CPU SSSP solution for source-distance
    if (!quick_mode)
    {
        if (!quiet_mode) { printf("Computing reference value ...\n"); }
        ReferenceSssp<VertexId, SizeT, Value, MARK_PREDECESSORS>(
            *graph,
            reference_check_label,
            reference_check_pred,
            src,
            quiet_mode);
        if (!quiet_mode) { printf("\n"); }
    }

    cpu_timer.Start();
    // Copy out results
    if (retval = util::GRError(problem->Extract(h_labels, h_preds),
        "SSSP Problem Data Extraction Failed", __FILE__, __LINE__))
        return retval;

    if (!quick_mode) {
        for (SizeT i = 0; i < graph->nodes; i++)
        {
            if (reference_check_label[i] == -1)
            {
                reference_check_label[i] = util::MaxValue<Value>();
            }
        }
    }

    if (!quiet_mode)
    {
        // Display Solution
        printf("\nFirst 40 labels of the GPU result.\n");
        DisplaySolution(h_labels, graph->nodes);
    }
    // Verify the result
    if (!quick_mode)
    {
        if (!quiet_mode) { printf("Label Validity: "); }
        int error_num = CompareResults(
                            h_labels, reference_check_label,
                            graph->nodes, true, quiet_mode);
        if (error_num > 0)
        {
            if (!quiet_mode) { printf("%d errors occurred.\n", error_num); }
        }
        if (!quiet_mode)
        {
            printf("\nFirst 40 labels of the reference CPU result.\n");
            DisplaySolution(reference_check_label, graph->nodes);
        }
    }

    info->ComputeTraversalStats(  // compute running statistics
        enactor->enactor_stats.GetPointer(), total_elapsed, h_labels);

    if (!quiet_mode)
    {
        if (MARK_PREDECESSORS)
        {
            printf("\nFirst 40 preds of the GPU result.\n");
            DisplaySolution(h_preds, graph->nodes);
            if (reference_check_label != NULL)
            {
                printf("\nFirst 40 preds of the reference CPU result (could be different because the paths are not unique).\n");
                DisplaySolution(reference_check_pred, graph->nodes);
            }
        }

        /*printf("\n\tMemory Usage(B)\t");
        for (int gpu = 0; gpu < num_gpus; gpu++)
            if (num_gpus > 1) {if (gpu != 0) printf(" #keys%d,0\t #keys%d,1\t #ins%d,0\t #ins%d,1", gpu, gpu, gpu, gpu); else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);}
            else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);
        if (num_gpus > 1) printf(" #keys%d", num_gpus);
        printf("\n");
        double max_queue_sizing_[2] = {0, 0}, max_in_sizing_ = 0;
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            size_t gpu_free, dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&gpu_free, &dummy);
            printf("GPU_%d\t %ld", gpu_idx[gpu], org_size[gpu] - gpu_free);
            for (int i = 0; i < num_gpus; i++)
            {
                for (int j = 0; j < 2; j++)
                {
                    SizeT x = problem->data_slices[gpu]->frontier_queues[i].keys[j].GetSize();
                    printf("\t %lld", (long long) x);
                    double factor = 1.0 * x / (num_gpus > 1 ? problem->graph_slices[gpu]->in_counter[i] : problem->graph_slices[gpu]->nodes);
                    if (factor > max_queue_sizing_[j]) max_queue_sizing_[j] = factor;
                }
                if (num_gpus > 1 && i != 0 )
                    for (int t = 0; t < 2; t++)
                    {
                        SizeT x = problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                        printf("\t %lld", (long long) x);
                        double factor = 1.0 * x / problem->graph_slices[gpu]->in_counter[i];
                        if (factor > max_in_sizing_) max_in_sizing_ = factor;
                    }
            }
            if (num_gpus > 1) printf("\t %lld", (long long)(problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize()));
            printf("\n");
        }
        printf("\t queue_sizing =\t %lf \t %lf", max_queue_sizing_[0], max_queue_sizing_[1]);
        if (num_gpus > 1) printf("\t in_sizing =\t %lf", max_in_sizing_);
        printf("\n");
        */
    }

    if (!quiet_mode)
    {
        Display_Memory_Usage(num_gpus, gpu_idx, org_size, problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
        Display_Performance_Profiling(enactor);
#endif
    }

    // Clean up
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    if (enactor         )
    {
        if (retval = util::GRError(enactor -> Release(),
            "BFS Enactor Release failed", __FILE__, __LINE__))
            return retval;
        delete   enactor         ; enactor          = NULL;
    }
    if (problem         )
    {
        if (retval = util::GRError(problem -> Release(),
            "BFS Problem Release failed", __FILE__, __LINE__))
            return retval;
        delete   problem         ; problem          = NULL;
    }
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
    if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}
    if (gpu_idx         ) {delete[] gpu_idx         ; gpu_idx          = NULL;}
    cpu_timer.Stop();
    info->info["postprocess_time"] = cpu_timer.ElapsedMillis();
    return retval;
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to info contains parameters and statistics.
 *
 * \return hipError_t object which indicates the success of
 * all CUDA function calls.
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value>
hipError_t RunTests_mark_predecessors(Info<VertexId, SizeT, Value> *info)
{
    if (info->info["mark_predecessors"].get_bool())
        return RunTests<VertexId, SizeT, Value, /*INSTRUMENT,
                 DEBUG, SIZE_CHECK,*/ true>(info);
    else
        return RunTests<VertexId, SizeT, Value, /*INSTRUMENT,
                 DEBUG, SIZE_CHECK,*/ false>(info);
}

/******************************************************************************
* Main
******************************************************************************/

template <
    typename VertexId,  // Use int as the vertex identifier
    typename SizeT,     // Use int as the graph size type
    typename Value>     // Use int as the value type
int main_(CommandLineArgs *args)
{
    CpuTimer cpu_timer, cpu_timer2;
    cpu_timer.Start();
    Csr <VertexId, SizeT, Value> csr(false);  // graph we process on
    Info<VertexId, SizeT, Value> *info = new Info<VertexId, SizeT, Value>;

    // graph construction or generation related parameters
    info->info["undirected"] = args -> CheckCmdLineFlag("undirected");
    info->info["edge_value"] = true;  // require per edge weight values
    info->info["random_edge_value"] = args -> CheckCmdLineFlag("random-edge-value");

    cpu_timer2.Start();
    info->Init("SSSP", *args, csr);  // initialize Info structure

    // force edge values to be 1, don't enable this unless you really want to
    //for (SizeT e=0; e < csr.edges; e++)
    //    csr.edge_values[e] = 1;
    cpu_timer2.Stop();
    info->info["load_time"] = cpu_timer2.ElapsedMillis();

    hipError_t retval = RunTests_mark_predecessors<VertexId, SizeT, Value>(info);  // run test
    cpu_timer.Stop();
    info->info["total_time"] = cpu_timer.ElapsedMillis();

    if (!(info->info["quiet_mode"].get_bool()))
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject
    if (info) {delete info; info=NULL;}
    return retval;
}

template <
    typename VertexId, // the vertex identifier type, usually int or long long
    typename SizeT   > // the size tyep, usually int or long long
int main_Value(CommandLineArgs *args)
{
// Disabled becaus atomicMin(long long*, long long) is not available
//    if (args -> CheckCmdLineFlag("64bit-Value"))
//        return main_<VertexId, SizeT, long long>(args);
//    else
        return main_<VertexId, SizeT, int      >(args);
}

template <
    typename VertexId>
int main_SizeT(CommandLineArgs *args)
{
// disabled to reduce compile time
    if (args -> CheckCmdLineFlag("64bit-SizeT"))
        return main_Value<VertexId, long long>(args);
    else
        return main_Value<VertexId, int      >(args);
}

int main_VertexId(CommandLineArgs *args)
{
    // disabled, because oprtr::filter::KernelPolicy::SmemStorage is too large for 64bit VertexId
    //if (args -> CheckCmdLineFlag("64bit-VertexId"))
    //    return main_SizeT<long long>(args);
    //else
        return main_SizeT<int      >(args);
}

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    return main_VertexId(&args);
}
// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
