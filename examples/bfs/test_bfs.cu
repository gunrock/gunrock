#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <algorithm>
#include <iostream>
#include <fstream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>
#include <gunrock/util/track_utils.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <gunrock/util/shared_utils.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        "test <graph-type> [graph-type-arguments]\n"
        "Graph type and graph type arguments:\n"
        "    market <matrix-market-file-name>\n"
        "        Reads a Matrix-Market coordinate-formatted graph of\n"
        "        directed/undirected edges from STDIN (or from the\n"
        "        optionally-specified file).\n"
        "    rmat (default: rmat_scale = 10, a = 0.57, b = c = 0.19)\n"
        "        Generate R-MAT graph as input\n"
        "        --rmat_scale=<vertex-scale>\n"
        "        --rmat_nodes=<number-nodes>\n"
        "        --rmat_edgefactor=<edge-factor>\n"
        "        --rmat_edges=<number-edges>\n"
        "        --rmat_a=<factor> --rmat_b=<factor> --rmat_c=<factor>\n"
        "        --rmat_seed=<seed>\n"
        "    rgg (default: rgg_scale = 10, rgg_thfactor = 0.55)\n"
        "        Generate Random Geometry Graph as input\n"
        "        --rgg_scale=<vertex-scale>\n"
        "        --rgg_nodes=<number-nodes>\n"
        "        --rgg_thfactor=<threshold-factor>\n"
        "        --rgg_threshold=<threshold>\n"
        "        --rgg_vmultipiler=<vmultipiler>\n"
        "        --rgg_seed=<seed>\n\n"
        "Optional arguments:\n"
        "[--device=<device_index>] Set GPU(s) for testing (Default: 0).\n"
        "[--undirected]            Treat the graph as undirected (symmetric).\n"
        "[--idempotence]           Whether or not to enable idempotent operation.\n"
        "[--instrumented]          Keep kernels statics [Default: Disable].\n"
        "                          total_queued, search_depth and barrier duty.\n"
        "                          (a relative indicator of load imbalance.)\n"
        "[--src=<Vertex-ID|largestdegree|randomize|randomize2|list>]\n"
        "                          Begins traversal from the source (Default: 0).\n"
        "                          If largestdegree: from largest degree vertex.\n"
        "                          If randomize: from a random source vertex.\n"
        "                          If randomize2: from a different random source vertex for each iteration.\n"
        "                          If list: need to provide a source list through --source_list=n0,n1,...,nk\n"
        "[--quick]                 Skip the CPU reference validation process.\n"
        "[--mark-pred]             Keep both label info and predecessor info.\n"
        "[--disable-size-check]    Disable frontier queue size check.\n"
        "[--grid-size=<grid size>] Maximum allowed grid size setting.\n"
        "[--queue-sizing=<factor>] Allocates a frontier queue sized at: \n"
        "                          (graph-edges * <factor>). (Default: 1.0)\n"
        "[--in-sizing=<in/out_queue_scale_factor>]\n"
        "                          Allocates a frontier queue sized at: \n"
        "                          (graph-edges * <factor>). (Default: 1.0)\n"
        "[--v]                     Print verbose per iteration debug info.\n"
        "[--iteration-num=<num>]   Number of runs to perform the test.\n"
        "[--traversal-mode=<0|1>]  Set traversal strategy, 0 for Load-Balanced\n"
        "                          1 for Dynamic-Cooperative (Default: dynamic\n"
        "                          determine based on average degree).\n"
        "[--partition-method=<random|biasrandom|clustered|metis>]\n"
        "                          Choose partitioner (Default use random).\n"
        "[--quiet]                 No output (unless --json is specified).\n"
        "[--json]                  Output JSON-format statistics to STDOUT.\n"
        "[--jsonfile=<name>]       Output JSON-format statistics to file <name>\n"
        "[--jsondir=<dir>]         Output JSON-format statistics to <dir>/name,\n"
        "                          where name is auto-generated.\n"
    );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @tparam VertexId
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 * @tparam ENABLE_IDEMPOTENCE
 *
 * @param[in] labels    Search depth from the source for each node.
 * @param[in] preds     Predecessor node id for each node.
 * @param[in] num_nodes Number of nodes in the graph.
 * @param[in] quiet     Don't print out anything to stdout
 */
template <
    typename VertexId,
    typename SizeT,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE >
void DisplaySolution(
    VertexId *labels,
    VertexId *preds,
    SizeT     num_nodes,
    bool quiet = false)
{
    if (quiet) { return; }
    // careful: if later code in this
    // function changes something, this
    // return is the wrong thing to do

    if (num_nodes > 40) { num_nodes = 40; }

    printf("\nFirst %lld labels of the GPU result:\n",
        (long long)num_nodes);

    printf("[");
    for (VertexId i = 0; i < num_nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(labels[i]);
        if (MARK_PREDECESSORS) //&& !ENABLE_IDEMPOTENCE)
        {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference BFS ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 * @tparam ENABLE_IDEMPOTENCE
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source_path Host-side vector to store CPU computed labels for each node
 * @param[in] predecessor Host-side vector to store CPU computed predecessor for each node
 * @param[in] src Source node where BFS starts
 * @param[in] quiet Don't print out anything to stdout
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE >
void ReferenceBFS(
    const Csr<VertexId, SizeT, Value> *graph,
    VertexId                          *source_path,
    VertexId                          *predecessor,
    VertexId                          src,
    bool                              quiet = false)
{
    // Initialize labels
    for (VertexId i = 0; i < graph->nodes; ++i)
    {
        source_path[i] = /*ENABLE_IDEMPOTENCE ? -1 :*/ util::MaxValue<VertexId>();
        if (MARK_PREDECESSORS)
        {
            predecessor[i] = util::InvalidValue<VertexId>();
        }
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    // Perform BFS
    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty())
    {
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        SizeT edges_begin = graph->row_offsets[dequeued_node];
        SizeT edges_end = graph->row_offsets[dequeued_node + 1];

        for (SizeT edge = edges_begin; edge < edges_end; ++edge)
        {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph->column_indices[edge];
            if (source_path[neighbor] > neighbor_dist) //|| source_path[neighbor] == -1)
            {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS)
                {
                    predecessor[neighbor] = dequeued_node;
                }
                if (search_depth < neighbor_dist)
                {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
    {
        predecessor[src] = util::InvalidValue<VertexId>();
    }

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    if (!quiet)
    {
        printf("CPU BFS finished in %lf msec. cpu_search_depth: %lld\n",
               elapsed, (long long)search_depth);
    }
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 * @tparam ENABLE_IDEMPOTENCE
 *
 * @param[in] info Pointer to info contains parameters and statistics.
 *
 * \return hipError_t object which indicates the success of
 * all CUDA function calls.
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value,
    bool        MARK_PREDECESSORS,
    bool        ENABLE_IDEMPOTENCE >
hipError_t RunTests(Info<VertexId, SizeT, Value> *info)
{
    typedef BFSProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS,
            ENABLE_IDEMPOTENCE>
            //(MARK_PREDECESSORS && ENABLE_IDEMPOTENCE) >
            Problem;  // does not use double buffer

    typedef BFSEnactor < Problem>
            //INSTRUMENT,
            //DEBUG,
            //SIZE_CHECK >
            Enactor;

    // parse configurations from mObject info
    Csr<VertexId, SizeT, Value> *graph = info->csr_ptr;
    Csr<VertexId, SizeT, Value> *inv_graph = info->csc_ptr;
    VertexId src                   = info->info["source_vertex"     ].get_int64();
    int      max_grid_size         = info->info["max_grid_size"     ].get_int  ();
    int      num_gpus              = info->info["num_gpus"          ].get_int  ();
    double   max_queue_sizing      = info->info["max_queue_sizing"  ].get_real ();
    double   max_queue_sizing1     = info->info["max_queue_sizing1" ].get_real ();
    double   max_in_sizing         = info->info["max_in_sizing"     ].get_real ();
    std::string partition_method   = info->info["partition_method"  ].get_str  ();
    double   partition_factor      = info->info["partition_factor"  ].get_real ();
    int      partition_seed        = info->info["partition_seed"    ].get_int  ();
    bool     quiet_mode            = info->info["quiet_mode"        ].get_bool ();
    bool     quick_mode            = info->info["quick_mode"        ].get_bool ();
    bool     stream_from_host      = info->info["stream_from_host"  ].get_bool ();
    std::string traversal_mode     = info->info["traversal_mode"    ].get_str  ();
    bool     instrument            = info->info["instrument"        ].get_bool ();
    bool     debug                 = info->info["debug_mode"        ].get_bool ();
    bool     size_check            = info->info["size_check"        ].get_bool ();
    int      iterations            = info->info["num_iteration"     ].get_int  ();
    std::string src_type           = info->info["source_type"       ].get_str  ();
    int      src_seed              = info->info["source_seed"       ].get_int  ();
    int      communicate_latency   = info->info["communicate_latency"].get_int ();
    float    communicate_multipy   = info->info["communicate_multipy"].get_real();
    int      expand_latency        = info->info["expand_latency"    ].get_int ();
    int      subqueue_latency      = info->info["subqueue_latency"  ].get_int ();
    int      fullqueue_latency     = info->info["fullqueue_latency" ].get_int ();
    int      makeout_latency       = info->info["makeout_latency"   ].get_int ();
    bool     direction_optimized   = info->info["direction_optimized"].get_bool();
    float    do_a                  = info->info["do_a"              ].get_real();
    float    do_b                  = info->info["do_b"              ].get_real();
    bool     undirected            = info->info["undirected"        ].get_bool();
    if (max_queue_sizing < 0) max_queue_sizing = 6.5;
    if (max_in_sizing < 0) max_in_sizing = 4;
    if (communicate_multipy > 1) max_in_sizing *= communicate_multipy;

    CpuTimer cpu_timer;
    hipError_t retval             = hipSuccess;

    cpu_timer.Start();
    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr   *context = (ContextPtr*)  info->context;
    hipStream_t *streams = (hipStream_t*)info->streams;

    // allocate host-side label array (for both reference and GPU results)
    VertexId *reference_labels      = new VertexId[graph->nodes];
    VertexId *reference_preds       = new VertexId[graph->nodes];
    VertexId *h_labels              = new VertexId[graph->nodes];
    VertexId *reference_check_label = (quick_mode) ? NULL : reference_labels;
    VertexId *reference_check_preds = NULL;
    VertexId *h_preds               = NULL;

    if (MARK_PREDECESSORS)
    {
        h_preds = new VertexId[graph->nodes];
        if (!quick_mode)
        {
            reference_check_preds = reference_preds;
        }
    }

    size_t *org_size = new size_t[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        if (retval = util::SetDevice(gpu_idx[gpu])) return retval;
        if (retval = util::GRError( hipMemGetInfo(&(org_size[gpu]), &dummy),
            "hipMemGetInfo failed", __FILE__, __LINE__)) return retval;
    }

    Problem* problem = new Problem(direction_optimized, undirected);  // allocate problem on GPU
    if (retval = util::GRError(problem->Init(
        stream_from_host,
        graph,
        inv_graph,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed),
        "BFS Problem Init failed", __FILE__, __LINE__)) return retval;

    Enactor* enactor = new Enactor(
        num_gpus, gpu_idx, instrument, debug, size_check, direction_optimized);  // enactor map
    if (retval = util::GRError(enactor->Init(
        context, problem, max_grid_size, traversal_mode),
        "BFS Enactor Init failed", __FILE__, __LINE__))
        return retval;

    enactor -> communicate_latency = communicate_latency;
    enactor -> communicate_multipy = communicate_multipy;
    enactor -> expand_latency      = expand_latency;
    enactor -> subqueue_latency    = subqueue_latency;
    enactor -> fullqueue_latency   = fullqueue_latency;
    enactor -> makeout_latency     = makeout_latency;
    enactor -> do_a                = do_a;
    enactor -> do_b                = do_b;

    if (retval = util::SetDevice(gpu_idx[0])) return retval;
    if (retval = util::latency::Test(
        streams[0], problem -> data_slices[0] -> latency_data,
        communicate_latency,
        communicate_multipy,
        expand_latency,
        subqueue_latency,
        fullqueue_latency,
        makeout_latency)) return retval;

    cpu_timer.Stop();
    info -> info["preprocess_time"] = cpu_timer.ElapsedMillis();

    // perform BFS
    double total_elapsed = 0.0;
    double single_elapsed = 0.0;
    double max_elapsed    = 0.0;
    double min_elapsed    = 1e10;
    json_spirit::mArray process_times;
    if (src_type == "random2")
    {
        if (src_seed == -1) src_seed = time(NULL);
        if (!quiet_mode)
            printf("src_seed = %d\n", src_seed);
        srand(src_seed);
    }
    if (!quiet_mode)
        printf("Using traversal-mode %s\n", traversal_mode.c_str());

    json_spirit::mArray source_list;
    if (src_type == "list")
        source_list = info->info["source_list"].get_array();
    for (int iter = 0; iter < iterations; ++iter)
    {
        if (src_type == "random2")
        {
            bool src_valid = false;
            while (!src_valid)
            {
                src = rand() % graph -> nodes;
                if (graph -> row_offsets[src] != graph -> row_offsets[src+1])
                    src_valid = true;
            }
        } else if (src_type == "list")
        {
            if (source_list.size() == 0)
            {
                if (!quiet_mode)
                    printf("No source list found. Use 0 as source.\n");
                src = 0;
            } else {
                src = source_list[iter].get_int();
            }
        }

        if (retval = util::GRError(problem->Reset(
            src, enactor->GetFrontierType(),
            max_queue_sizing, max_queue_sizing1),
            "BFS Problem Reset failed", __FILE__, __LINE__))
            return retval;

        if (retval = util::GRError(enactor->Reset(),
            "BFS Enactor Reset failed", __FILE__, __LINE__))
            return retval;

        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            if (retval = util::SetDevice(gpu_idx[gpu]))
                return retval;
            if (retval = util::GRError(hipDeviceSynchronize(),
                "hipDeviceSynchronize failed", __FILE__, __LINE__))
                return retval;
        }

        if (!quiet_mode)
        {
            printf("__________________________\n"); fflush(stdout);
        }

        cpu_timer.Start();
        if (retval = util::GRError(enactor->Enact(src, traversal_mode),
            "BFS Enact failed", __FILE__, __LINE__)) return retval;
        cpu_timer.Stop();
        single_elapsed = cpu_timer.ElapsedMillis();
        total_elapsed += single_elapsed;
        process_times.push_back(single_elapsed);
        if (single_elapsed > max_elapsed) max_elapsed = single_elapsed;
        if (single_elapsed < min_elapsed) min_elapsed = single_elapsed;
        if (!quiet_mode)
        {
            printf("--------------------------\n"
                "iteration %d elapsed: %lf ms, src = %lld, #iteration = %lld\n",
                iter, single_elapsed, (long long)src,
                (long long)enactor -> enactor_stats -> iteration);
            fflush(stdout);
        }
    }
    total_elapsed /= iterations;
    info -> info["process_times"] = process_times;
    info -> info["min_process_time"] = min_elapsed;
    info -> info["max_process_time"] = max_elapsed;

    // compute reference CPU BFS solution for source-distance
    if (!quick_mode)
    {
        if (!quiet_mode)
        {
            printf("Computing reference value ...\n");
        }
        ReferenceBFS<VertexId, SizeT, Value,
            MARK_PREDECESSORS, ENABLE_IDEMPOTENCE>(
            graph,
            reference_check_label,
            reference_check_preds,
            src,
            quiet_mode);
        if (!quiet_mode)
        {
            printf("\n");
        }
    }

    cpu_timer.Start();
    // copy out results
    if (retval = util::GRError(problem->Extract(h_labels, h_preds),
        "BFS Problem Extraction failed", __FILE__, __LINE__)) return retval;

    // verify the result
    if ((!quick_mode) && (!quiet_mode))
    {
        printf("Label Validity: ");
        int num_errors = CompareResults(
            h_labels, reference_check_label,
            graph->nodes, true, quiet_mode);
        if (num_errors > 0)
        {
            printf("%d errors occurred.", num_errors);
        }
        printf("\n");

        if (MARK_PREDECESSORS)
        {
            printf("Predecessor Validity: \n");
            num_errors = 0;
            #pragma omp parallel for
            for (VertexId v=0; v<graph->nodes; v++)
            {
                if (h_labels[v] ==
                    /*(ENABLE_IDEMPOTENCE ? -1 :*/ util::MaxValue<VertexId>())
                    continue; // unvisited vertex
                if (v == src && h_preds[v] == util::InvalidValue<VertexId>()) continue; // source vertex
                VertexId pred = h_preds[v];
                if (pred >= graph->nodes || pred < 0)
                {
                    if (num_errors == 0)
                        printf("INCORRECT: pred[%lld] : %lld out of bound\n",
                            (long long)v, (long long)pred);
                    #pragma omp atomic
                    num_errors ++;
                    continue;
                }
                if (h_labels[v] != h_labels[pred] + 1)
                {
                    if (num_errors == 0)
                        printf("INCORRECT: label[%lld] (%lld) != label[%lld] (%lld) + 1\n",
                            (long long)v, (long long)h_labels[v], (long long)pred, (long long)h_labels[pred]);
                    #pragma omp atomic
                    num_errors ++;
                    continue;
                }

                bool v_found = false;
                for (SizeT t = graph->row_offsets[pred]; t < graph->row_offsets[pred+1]; t++)
                if (v == graph->column_indices[t])
                {
                    v_found = true;
                    break;
                }
                if (!v_found)
                {
                    if (num_errors == 0)
                        printf("INCORRECT: Vertex %lld not in Vertex %lld's neighbor list\n",
                            (long long)v, (long long)pred);
                    #pragma omp atomic
                    num_errors ++;
                    continue;
                }
            }

            if (num_errors > 0)
            {
                printf("%d errors occurred.", num_errors);
            } else printf("CORRECT");
            printf("\n");
        }

    }

    if (!quick_mode && TO_TRACK)
    {
        VertexId **v_ = NULL;
        if (num_gpus > 1)
        {
            v_ = new VertexId*[num_gpus];
            for (int gpu=0; gpu<num_gpus; gpu++)
            {
                v_[gpu] = new VertexId[graph->nodes];
                for (VertexId v=0; v<graph->nodes; v++)
                    v_[gpu][v] = -1;
                for (VertexId v=0; v<problem->sub_graphs[gpu].nodes; v++)
                    v_[gpu][problem->original_vertexes[gpu][v]] = v;
            }
        }
        util::Track_Results(graph, num_gpus, (VertexId)1, h_labels, reference_check_label,
            num_gpus > 1 ? problem->partition_tables[0] : NULL, v_);
        char file_name[512];
        sprintf(file_name, "./eval/error_dump/error_%lld_%d.txt", (long long)time(NULL), gpu_idx[0]);
        util::Output_Errors(file_name, graph -> nodes, num_gpus, (VertexId)0, h_labels, reference_check_label,
            num_gpus > 1 ? problem->partition_tables[0] : NULL, v_);
        if (num_gpus > 1)
        {
            for (int gpu=0; gpu<num_gpus; gpu++)
            {
                delete[] v_[gpu]; v_[gpu] = NULL;
            }
            delete[] v_; v_=NULL;
        }
    }

    // display Solution
    if (!quiet_mode)
    {
        DisplaySolution<VertexId, SizeT, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE>
        (h_labels, h_preds, graph->nodes, quiet_mode);
    }

    info->ComputeTraversalStats(  // compute running statistics
        enactor->enactor_stats.GetPointer(), total_elapsed, h_labels);

    if (!quiet_mode)
    {
        Display_Memory_Usage(num_gpus, gpu_idx, org_size, problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
        Display_Performance_Profiling(enactor);
#endif
    }

    // Clean up
    if (enactor         )
    {
        if (retval = util::GRError(enactor -> Release(),
            "BFS Enactor Release failed", __FILE__, __LINE__))
            return retval;
        delete   enactor         ; enactor          = NULL;
    }
    if (problem         )
    {
        if (retval = util::GRError(problem -> Release(),
            "BFS Problem Release failed", __FILE__, __LINE__))
            return retval;
        delete   problem         ; problem          = NULL;
    }
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (gpu_idx         ) {delete[] gpu_idx         ; gpu_idx          = NULL;}
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    cpu_timer.Stop();
    info->info["postprocess_time"] = cpu_timer.ElapsedMillis();

    if (h_preds         )
    {
        if (info->info["output_filename"].get_str() != "")
        {
            cpu_timer.Start();
            std::ofstream fout;
            size_t buf_size = 1024 * 1024 * 16;
            char *fout_buf = new char[buf_size];
            fout.rdbuf() -> pubsetbuf(fout_buf, buf_size);
            fout.open(info->info["output_filename"].get_str().c_str());

            for (VertexId v=0; v<graph->nodes; v++)
            {
                if (v == src) fout<< v+1 << "," << v+1 << std::endl; // root node
                else if (h_preds[v] != -2) // valid pred
                    fout<< v+1 << "," << h_preds[v]+1 << std::endl;
            }

            fout.close();
            delete[] fout_buf; fout_buf = NULL;
            cpu_timer.Stop();
            info->info["write_time"] = cpu_timer.ElapsedMillis();
        }
        delete[] h_preds         ; h_preds          = NULL;
    }
    return retval;
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] info Pointer to info contains parameters and statistics.
 *
 * \return hipError_t object which indicates the success of
 * all CUDA function calls.
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value,
    bool        MARK_PREDECESSORS >
hipError_t RunTests_enable_idempotence(Info<VertexId, SizeT, Value> *info)
{
    if (info->info["idempotent"].get_bool())
        return RunTests <VertexId, SizeT, Value,/* INSTRUMENT, DEBUG, SIZE_CHECK,*/
                 MARK_PREDECESSORS, true > (info);
    else
        return RunTests <VertexId, SizeT, Value,/* INSTRUMENT, DEBUG, SIZE_CHECK,*/
                 MARK_PREDECESSORS, false> (info);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to info contains parameters and statistics.
 *
 * \return hipError_t object which indicates the success of
 * all CUDA function calls.
 */
template <
    typename    VertexId,
    typename    SizeT,
    typename    Value>
hipError_t RunTests_mark_predecessors(Info<VertexId, SizeT, Value> *info)
{
    if (info->info["mark_predecessors"].get_bool())
        return RunTests_enable_idempotence<VertexId, SizeT, Value, /*INSTRUMENT,
                                    DEBUG, SIZE_CHECK,*/  true> (info);
    else
        return RunTests_enable_idempotence<VertexId, SizeT, Value,/* INSTRUMENT,
                                    DEBUG, SIZE_CHECK,*/ false> (info);
}

/******************************************************************************
* Main
******************************************************************************/

template <
    typename VertexId,  // use int as the vertex identifier
    typename SizeT   ,  // use int as the graph size type
    typename Value   >  // use int as the value type
int main_(CommandLineArgs *args)
{
    CpuTimer cpu_timer, cpu_timer2;
    cpu_timer.Start();
    //typedef int VertexId;  // Use int as the vertex identifier
    //typedef int Value;     // Use int as the value type
    //typedef long long SizeT;     // Use int as the graph size type

    Csr<VertexId, SizeT, Value> csr(false);  // CSR graph we process on
    Csr<VertexId, SizeT, Value> csc(false);  // CSC graph we process on
    Info<VertexId, SizeT, Value> *info = new Info<VertexId, SizeT, Value>;

    // graph construction or generation related parameters
    info->info["undirected"] = args -> CheckCmdLineFlag("undirected");

    cpu_timer2.Start();
    info->Init("BFS", *args, csr, csc);  // initialize Info structure
    cpu_timer2.Stop();
    info->info["load_time"] = cpu_timer2.ElapsedMillis();

    hipError_t retval = RunTests_mark_predecessors<VertexId, SizeT, Value>(info);  // run test

    cpu_timer.Stop();
    info->info["total_time"] = cpu_timer.ElapsedMillis();

    if (!(info->info["quiet_mode"].get_bool()))
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject
    if (info) {delete info; info=NULL;}
    return retval;
}

template <
    typename VertexId, // the vertex identifier type, usually int or long long
    typename SizeT   > // the size tyep, usually int or long long
int main_Value(CommandLineArgs *args)
{
    // Value = VertexId for bfs
    return main_<VertexId, SizeT, VertexId>(args);
//    if (args -> CheckCmdLineFlag("64bit-Value"))
//        return main_<VertexId, SizeT, long long>(args);
//    else
//        return main_<VertexId, SizeT, int      >(args);
}

template <
    typename VertexId>
int main_SizeT(CommandLineArgs *args)
{
// can be disabled to reduce compile time
    if (args -> CheckCmdLineFlag("64bit-SizeT") || sizeof(VertexId) > 4)
        return main_Value<VertexId, long long>(args);
    else
        return main_Value<VertexId, int      >(args);
}

int main_VertexId(CommandLineArgs *args)
{
// can be disabled to reduce compile time
// atomicMin(long long) is only available for compute capability 3.5 or higher
    if (args -> CheckCmdLineFlag("64bit-VertexId"))
//#if __GR_CUDA_ARCH__ <= 300
//    {
//        printf("64bit-VertexId disabled, because atomicMin(long long) is only supported by compute capability 3.5 or higher\n");
//        return 1;
//    }
//#else
        return main_SizeT<long long>(args);
//#endif
    else
        return main_SizeT<int      >(args);
}

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    return main_VertexId(&args);
}
// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
