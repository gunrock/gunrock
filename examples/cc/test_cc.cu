// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_cc.cu
 *
 * @brief Simple test driver program for connected component.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// CC includes
#include <gunrock/app/cc/cc_enactor.cuh>
#include <gunrock/app/cc/cc_problem.cuh>
#include <gunrock/app/cc/cc_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

// Boost includes for CPU CC reference algorithms
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/connected_components.hpp>

#include <gunrock/util/shared_utils.cuh>

using namespace gunrock;
using namespace gunrock::app;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::cc;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

template <typename VertexId, typename SizeT>
struct CcList
{
    VertexId root;
    SizeT    histogram;

    CcList(VertexId root, SizeT histogram) :
        root(root), histogram(histogram) {}
};

template<typename CcList>
bool CCCompare(
    CcList elem1,
    CcList elem2)
{
    return elem1.histogram > elem2.histogram;
}

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        "test <graph-type> [graph-type-arguments]\n"
        "Graph type and graph type arguments:\n"
        "    market <matrix-market-file-name>\n"
        "        Reads a Matrix-Market coordinate-formatted graph of\n"
        "        directed/undirected edges from STDIN (or from the\n"
        "        optionally-specified file).\n"
        "    rmat (default: rmat_scale = 10, a = 0.57, b = c = 0.19)\n"
        "        Generate R-MAT graph as input\n"
        "        --rmat_scale=<vertex-scale>\n"
        "        --rmat_nodes=<number-nodes>\n"
        "        --rmat_edgefactor=<edge-factor>\n"
        "        --rmat_edges=<number-edges>\n"
        "        --rmat_a=<factor> --rmat_b=<factor> --rmat_c=<factor>\n"
        "        --rmat_seed=<seed>\n"
        "    rgg (default: rgg_scale = 10, rgg_thfactor = 0.55)\n"
        "        Generate Random Geometry Graph as input\n"
        "        --rgg_scale=<vertex-scale>\n"
        "        --rgg_nodes=<number-nodes>\n"
        "        --rgg_thfactor=<threshold-factor>\n"
        "        --rgg_threshold=<threshold>\n"
        "        --rgg_vmultipiler=<vmultipiler>\n"
        "        --rgg_seed=<seed>\n\n"
        "Optional arguments:\n"
        "[--device=<device_index>] Set GPU(s) for testing (Default: 0).\n"
        "[--instrumented]          Keep kernels statics [Default: Disable].\n"
        "                          total_queued, search_depth and barrier duty.\n"
        "                          (a relative indicator of load imbalance.)\n"
        "[--quick]                 Skip the CPU reference validation process.\n"
        "[--disable-size-check]    Disable frontier queue size check.\n"
        "[--grid-size=<grid size>] Maximum allowed grid size setting.\n"
        "[--queue-sizing=<factor>] Allocates a frontier queue sized at: \n"
        "                          (graph-edges * <factor>). (Default: 1.0)\n"
        "[--in-sizing=<in/out_queue_scale_factor>]\n"
        "                          Allocates a frontier queue sized at: \n"
        "                          (graph-edges * <factor>). (Default: 1.0)\n"
        "[--v]                     Print verbose per iteration debug info.\n"
        "[--iteration-num=<num>]   Number of runs to perform the test.\n"
        "[--partition-method=<random|biasrandom|clustered|metis>]\n"
        "                          Choose partitioner (Default use random).\n"
        "[--quiet]                 No output (unless --json is specified).\n"
        "[--json]                  Output JSON-format statistics to STDOUT.\n"
        "[--jsonfile=<name>]       Output JSON-format statistics to file <name>\n"
        "[--jsondir=<dir>]         Output JSON-format statistics to <dir>/name,\n"
        "                          where name is auto-generated.\n"
    );
}

/**
 * @brief Displays the CC result (i.e., number of components)
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] comp_ids Host-side vector to store computed component id for each node
 * @param[in] nodes Number of nodes in the graph
 * @param[in] num_components Number of connected components in the graph
 * @param[in] roots Host-side vector stores the root for each node in the graph
 * @param[in] histogram Histogram of connected component ids
 */
template<typename VertexId, typename SizeT>
void DisplaySolution(
    VertexId     *comp_ids,
    SizeT        nodes,
    SizeT        num_components,
    VertexId     *roots,
    SizeT        *histogram)
{
    typedef CcList<VertexId, SizeT> CcListType;
    //printf("Number of components: %d\n", num_components);

    if (nodes <= 40)
    {
        printf("[");
        for (VertexId i = 0; i < nodes; ++i)
        {
            PrintValue(i);
            printf(":");
            PrintValue(comp_ids[i]);
            printf(",");
            printf(" ");
        }
        printf("]\n");
    }
    else
    {
        //sort the components by size
        CcListType *cclist =
            (CcListType*)malloc(sizeof(CcListType) * num_components);
        for (SizeT i = 0; i < num_components; ++i)
        {
            cclist[i].root = roots[i];
            cclist[i].histogram = histogram[i];
        }
        std::stable_sort(
            cclist, cclist + num_components, CCCompare<CcListType>);

        // Print out at most top 10 largest components
        SizeT top = (num_components < 10) ? num_components : 10;
        printf("Top %lld largest components:\n", (long long)top);
        for (SizeT i = 0; i < top; ++i)
        {
            printf("CC ID: %lld, CC Root: %lld, CC Size: %lld\n",
                   (long long)i, (long long)cclist[i].root, (long long)cclist[i].histogram);
        }

        free(cclist);
    }
}

/******************************************************************************
 * CC Testing Routines
 *****************************************************************************/

/**
 * @brief CPU-based reference CC algorithm using Boost Graph Library
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in]  graph  Reference to the CSR graph we process on
 * @param[out] labels Host-side vector to store the component id for each node in the graph
 * @param[in] quiet Don't print out anything to stdout
 *
 * \return Number of connected components in the graph
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value >
unsigned int ReferenceCC(
    const Csr<VertexId, SizeT, Value> &graph,
    VertexId *labels,
    bool quiet = false)
{
    using namespace boost;
    SizeT    *row_offsets    = graph.row_offsets;
    VertexId *column_indices = graph.column_indices;
    SizeT     num_nodes      = graph.nodes;

    typedef adjacency_list <vecS, vecS, undirectedS> Graph;
    Graph G;
    for (int i = 0; i < num_nodes; ++i)
    {
        for (int j = row_offsets[i]; j < row_offsets[i + 1]; ++j)
        {
            add_edge(i, column_indices[j], G);
        }
    }
    CpuTimer cpu_timer;
    cpu_timer.Start();
    SizeT num_components = connected_components(G, &labels[0]);
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    if (!quiet) { printf("CPU CC finished in %lf msec.\n", elapsed); }
    return num_components;
}

/**
 * @brief Convert component IDs.
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] labels
 * @param[in] num_nodes
 * @param[in] num_components
 */
template <
    typename VertexId,
    typename SizeT >
void ConvertIDs(
    VertexId *labels,
    SizeT    num_nodes,
    SizeT    num_components)
{
    VertexId *min_nodes = new VertexId[num_nodes];

    for (int cc = 0; cc < num_nodes; cc++)
        min_nodes[cc] = num_nodes;
    for (int node = 0; node < num_nodes; node++)
        if (min_nodes[labels[node]] > node) min_nodes[labels[node]] = node;
    for (int node = 0; node < num_nodes; node++)
        labels[node] = min_nodes[labels[node]];
    delete[] min_nodes; min_nodes = NULL;
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] info Pointer to info contains parameters and statistics.
 *
 * \return hipError_t object which indicates the success of
 * all CUDA function calls.
 */
template <
    typename VertexId,
    typename SizeT,
    typename Value>
hipError_t RunTests(Info<VertexId, SizeT, Value> *info)
{
    typedef CCProblem < VertexId,
            SizeT,
            Value> Problem;  // use double buffer for advance and filter

    typedef CCEnactor < Problem>
            //INSTRUMENT,
            //DEBUG,
            //SIZE_CHECK >
            Enactor;

    // parse configurations from mObject info
    Csr<VertexId, SizeT, Value> *graph = info->csr_ptr;
    int     max_grid_size          = info->info["max_grid_size"     ].get_int  ();
    int     num_gpus               = info->info["num_gpus"          ].get_int  ();
    double  max_queue_sizing       = info->info["max_queue_sizing"  ].get_real ();
    double  max_queue_sizing1      = info->info["max_queue_sizing1" ].get_real ();
    double  max_in_sizing          = info->info["max_in_sizing"     ].get_real ();
    std::string partition_method   = info->info["partition_method"  ].get_str  ();
    double  partition_factor       = info->info["partition_factor"  ].get_real ();
    int     partition_seed         = info->info["partition_seed"    ].get_int  ();
    bool    quiet_mode             = info->info["quiet_mode"        ].get_bool ();
    bool    quick_mode             = info->info["quick_mode"        ].get_bool ();
    bool    stream_from_host       = info->info["stream_from_host"  ].get_bool ();
    std::string traversal_mode     = info->info["traversal_mode"    ].get_str  ();
    bool    instrument             = info->info["instrument"        ].get_bool ();
    bool    debug                  = info->info["debug_mode"        ].get_bool ();
    bool    size_check             = info->info["size_check"        ].get_bool ();
    int     iterations             = info->info["num_iteration"     ].get_int();
    int     communicate_latency    = info->info["communicate_latency"].get_int ();
    float   communicate_multipy    = info->info["communicate_multipy"].get_real();
    int     expand_latency         = info->info["expand_latency"    ].get_int ();
    int     subqueue_latency       = info->info["subqueue_latency"  ].get_int ();
    int     fullqueue_latency      = info->info["fullqueue_latency" ].get_int ();
    int     makeout_latency        = info->info["makeout_latency"   ].get_int ();
    if (max_queue_sizing < 0) max_queue_sizing = 1.0;
    if (max_in_sizing < 0) max_in_sizing = 1.1;
    if (communicate_multipy > 1) max_in_sizing *= communicate_multipy;
    CpuTimer cpu_timer;
    hipError_t retval;

    cpu_timer.Start();
    json_spirit::mArray device_list = info->info["device_list"].get_array();
    int* gpu_idx = new int[num_gpus];
    for (int i = 0; i < num_gpus; i++) gpu_idx[i] = device_list[i].get_int();

    // TODO: remove after merge mgpu-cq
    ContextPtr   *context = (ContextPtr*)  info->context;
    hipStream_t *streams = (hipStream_t*)info->streams;

    // Allocate host-side array (for both reference and GPU-computed results)
    VertexId    *reference_component_ids = new VertexId[graph->nodes];
    VertexId    *h_component_ids        = new VertexId[graph->nodes];
    VertexId    *reference_check        = (quick_mode) ? NULL : reference_component_ids;
    SizeT        ref_num_components     = 0;

    //printf("0: node %d: %d -> %d, node %d: %d -> %d\n", 131070, graph->row_offsets[131070], graph->row_offsets[131071], 131071, graph->row_offsets[131071], graph->row_offsets[131072]);
    //for (int edge = 0; edge < graph->edges; edge ++)
    //{
    //    if (graph->column_indices[edge] == 131070 || graph->column_indices[edge] == 131071)
    //    printf("edge %d: -> %d\n", edge, graph->column_indices[edge]);
    //}

    //util::cpu_mt::PrintCPUArray("row_offsets", graph->row_offsets, graph->nodes+1);
    //util::cpu_mt::PrintCPUArray("colunm_indices", graph->column_indices, graph->edges);
    size_t *org_size = new size_t[num_gpus];
    for (int gpu = 0; gpu < num_gpus; gpu++)
    {
        size_t dummy;
        if (retval = util::SetDevice(gpu_idx[gpu])) return retval;
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    Problem* problem = new Problem;  // allocate problem on GPU
    if (retval = util::GRError(problem->Init(
        stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed),
        "CC Problem Initialization Failed", __FILE__, __LINE__))
        return retval;

    Enactor* enactor = new Enactor(
        num_gpus, gpu_idx, instrument, debug, size_check);  // enactor map
    if (retval = util::GRError(enactor->Init(
        context, problem, traversal_mode, max_grid_size),
        "CC Enactor Init failed", __FILE__, __LINE__))
        return retval;

    enactor -> communicate_latency = communicate_latency;
    enactor -> communicate_multipy = communicate_multipy;
    enactor -> expand_latency      = expand_latency;
    enactor -> subqueue_latency    = subqueue_latency;
    enactor -> fullqueue_latency   = fullqueue_latency;
    enactor -> makeout_latency     = makeout_latency;

    if (retval = util::SetDevice(gpu_idx[0])) return retval;
    if (retval = util::latency::Test(
        streams[0], problem -> data_slices[0] -> latency_data,
        communicate_latency,
        communicate_multipy,
        expand_latency,
        subqueue_latency,
        fullqueue_latency,
        makeout_latency)) return retval;

    cpu_timer.Stop();
    info -> info["preprocess_time"] = cpu_timer.ElapsedMillis();

    // compute reference CPU CC
    if (!quick_mode)
    {
        if (!quiet_mode) { printf("Computing reference value ...\n"); }
        ref_num_components = ReferenceCC(*graph, reference_check, quiet_mode);
        if (!quiet_mode) { printf("\n"); }
    }

    // perform CC
    double total_elapsed = 0.0;
    double single_elapsed = 0.0;
    double max_elapsed    = 0.0;
    double min_elapsed    = 1e10;
    json_spirit::mArray process_times;
    if (!quiet_mode) printf("Using traversal mode %s\n", traversal_mode.c_str());
    for (SizeT iter = 0; iter < iterations; ++iter)
    {
        if (retval = util::GRError(problem->Reset(
            enactor->GetFrontierType(), max_queue_sizing),
            "CC Problem Data Reset Failed", __FILE__, __LINE__))
            return retval;
        if (retval = util::GRError(enactor->Reset(),
            "CC Enactor Reset failed", __FILE__, __LINE__))
            return retval;

        if (!quiet_mode)
        {
            printf("_________________________\n"); fflush(stdout);
        }
        cpu_timer.Start();
        if (retval = util::GRError(enactor->Enact(traversal_mode),
            "CC Problem Enact Failed", __FILE__, __LINE__))
            return retval;
        cpu_timer.Stop();
        single_elapsed = cpu_timer.ElapsedMillis();
        total_elapsed += single_elapsed;
        process_times.push_back(single_elapsed);
        if (single_elapsed > max_elapsed) max_elapsed = single_elapsed;
        if (single_elapsed < min_elapsed) min_elapsed = single_elapsed;
        if (!quiet_mode)
        {
            printf("-------------------------\n"
                "iteration %lld elapsed: %lf ms\n",
                (long long)iter, single_elapsed);
            fflush(stdout);
        }

    }
    total_elapsed /= iterations;
    info -> info["process_times"] = process_times;
    info -> info["min_process_time"] = min_elapsed;
    info -> info["max_process_time"] = max_elapsed;

    cpu_timer.Start();
    // copy out results
    if (retval = util::GRError(problem->Extract(h_component_ids),
        "CC Problem Data Extraction Failed", __FILE__, __LINE__))
        return retval;

    // validity
    if (!quick_mode)
    {
        if (ref_num_components == problem->num_components)
        {
            if (!quiet_mode)
            {
                printf("CORRECT. Component Count: %lld\n", (long long)ref_num_components);
            }
        }
        else
        {
            if (!quiet_mode)
            {
                printf(
                    "INCORRECT. Ref Component Count: %lld, "
                    "GPU Computed Component Count: %lld\n",
                    (long long)ref_num_components, (long long)problem->num_components);
            }
        }
    }
    else
    {
        if (!quiet_mode)
        {
            printf("Component Count: %lld\n", (long long) problem->num_components);
        }
    }
    if (!quick_mode)
    {
        ConvertIDs<VertexId, SizeT>(reference_check, graph->nodes, ref_num_components);
        ConvertIDs<VertexId, SizeT>(h_component_ids, graph->nodes, problem->num_components);
        if (!quiet_mode)
        {
            printf("Label Validity: ");
        }
        SizeT error_num = CompareResults(
            h_component_ids, reference_check, graph->nodes, true, quiet_mode);
        if (error_num > 0)
        {
            if (!quiet_mode) { printf("%lld errors occurred.\n", (long long)error_num); }
        }
        else
        {
            if (!quiet_mode) { printf("\n"); }
        }
    }

    //if (ref_num_components == csr_problem->num_components)
    {
        // Compute size and root of each component
        VertexId *h_roots      = new VertexId[problem->num_components];
        SizeT    *h_histograms = new SizeT   [problem->num_components];

        //printf("num_components = %d\n", problem->num_components);
        problem->ComputeCCHistogram(h_component_ids, h_roots, h_histograms);
        //printf("num_components = %d\n", problem->num_components);

        if (!quiet_mode)
        {
            // Display Solution
            DisplaySolution(h_component_ids, graph->nodes,
                problem->num_components, h_roots, h_histograms);
        }

        if (h_roots     ) {delete[] h_roots     ; h_roots      = NULL;}
        if (h_histograms) {delete[] h_histograms; h_histograms = NULL;}
    }

    info->ComputeCommonStats(  // compute running statistics
        enactor->enactor_stats.GetPointer(), total_elapsed, h_component_ids, true);

    if (!quiet_mode)
    {
        Display_Memory_Usage(num_gpus, gpu_idx, org_size, problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
        Display_Performance_Profiling(enactor);
#endif
    }

    /*if (!quiet_mode)
    {
        printf("\n\tMemory Usage(B)\t");
        for (int gpu = 0; gpu < num_gpus; gpu++)
            if (num_gpus > 1)
            {
                if (gpu != 0) printf(" #keys%d\t #ins%d,0\t #ins%d,1", gpu, gpu, gpu);
                else printf(" $keys%d", gpu);
            }
            else printf(" #keys%d", gpu);
        if (num_gpus > 1) printf(" #keys%d", num_gpus);
        printf("\n");

        double max_key_sizing = 0, max_in_sizing_ = 0;
        for (int gpu = 0; gpu < num_gpus; gpu++)
        {
            size_t gpu_free, dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&gpu_free, &dummy);
            printf("GPU_%d\t %ld", gpu_idx[gpu], org_size[gpu] - gpu_free);
            for (int i = 0; i < num_gpus; i++)
            {
                SizeT x = problem->data_slices[gpu]->frontier_queues[i].keys[0].GetSize();
                printf("\t %lld", (long long)x);
                double factor = 1.0 * x / (num_gpus > 1 ? problem->graph_slices[gpu]->in_counter[i] : problem->graph_slices[gpu]->nodes);
                if (factor > max_key_sizing) max_key_sizing = factor;
                if (num_gpus > 1 && i != 0 )
                    for (int t = 0; t < 2; t++)
                    {
                        x = problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                        printf("\t %lld", (long long)x);
                        factor = 1.0 * x / problem->graph_slices[gpu]->in_counter[i];
                        if (factor > max_in_sizing_) max_in_sizing_ = factor;
                    }
            }
            if (num_gpus > 1) printf("\t %lld", (long long)problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize());
            printf("\n");
        }
        printf("\t key_sizing =\t %lf", max_key_sizing);
        if (num_gpus > 1) printf("\t in_sizing =\t %lf", max_in_sizing_);
        printf("\n");
    }*/

    // Cleanup
    if (org_size               ) {delete[] org_size               ; org_size                = NULL;}
    if (problem                ) {delete   problem                ; problem                 = NULL;}
    if (enactor                ) {delete   enactor                ; enactor                 = NULL;}
    if (reference_component_ids) {delete[] reference_component_ids; reference_component_ids = NULL;}
    if (h_component_ids        ) {delete[] h_component_ids        ; h_component_ids         = NULL;}
    if (gpu_idx                ) {delete[] gpu_idx                ; gpu_idx                 = NULL;}
    cpu_timer.Stop();
    info->info["postprocess_time"] = cpu_timer.ElapsedMillis();
    return retval;
}

/******************************************************************************
 * Main
 ******************************************************************************/
template <
    typename VertexId,  // use int as the vertex identifier
    typename SizeT   ,  // use int as the graph size type
    typename Value   >  // use int as the value type
int main_(CommandLineArgs *args)
{
    CpuTimer cpu_timer, cpu_timer2;
    cpu_timer.Start();
    Csr <VertexId, SizeT, Value> csr(false);  // graph we process on
    Info<VertexId, SizeT, Value> *info = new Info<VertexId, SizeT, Value>;

    // graph construction or generation related parameters
    info->info["undirected"] = true;   // require undirected input graph

    cpu_timer2.Start();
    info->Init("CC", *args, csr);  // initialize Info structure
    graphio::RemoveStandaloneNodes<VertexId, SizeT, Value>(
        &csr, args->CheckCmdLineFlag("quiet"));
        cpu_timer2.Stop();
    info->info["load_time"] = cpu_timer2.ElapsedMillis();

    RunTests<VertexId, SizeT, Value>(info);  // run test

    cpu_timer.Stop();
    info->info["total_time"] = cpu_timer.ElapsedMillis();

    if (!(info->info["quiet_mode"].get_bool()))
    {
        info->DisplayStats();  // display collected statistics
    }

    info->CollectInfo();  // collected all the info and put into JSON mObject
    if (info) {delete info; info=NULL;}
    return 0;
}

template <
    typename VertexId, // the vertex identifier type, usually int or long long
    typename SizeT   > // the size tyep, usually int or long long
int main_Value(CommandLineArgs *args)
{
//    if (args -> CheckCmdLineFlag("64bit-Value"))
//        return main_<VertexId, SizeT, long long>(args);
//    else
//        return main_<VertexId, SizeT, int      >(args);
      return main_<VertexId, SizeT, VertexId>(args); // Value = VertexId for CC
}

template <
    typename VertexId>
int main_SizeT(CommandLineArgs *args)
{
// disabled to reduce compile time
    if (args -> CheckCmdLineFlag("64bit-SizeT"))
        return main_Value<VertexId, long long>(args);
    else
        return main_Value<VertexId, int      >(args);
}

int main_VertexId(CommandLineArgs *args)
{
    // disabled, because oprtr::filter::KernelPolicy::SmemStorage is too large for 64bit VertexId
    //if (args -> CheckCmdLineFlag("64bit-VertexId"))
    //    return main_SizeT<long long>(args);
    //else
        return main_SizeT<int      >(args);
}

int main(int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int graph_args = argc - args.ParsedArgc() - 1;
    if (argc < 2 || graph_args < 1 || args.CheckCmdLineFlag("help"))
    {
        Usage();
        return 1;
    }

    return main_VertexId(&args);
}
// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
