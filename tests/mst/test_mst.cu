// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_mst.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <utility>
#include <iostream>
#include <cstdlib>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// MST includes
#include <gunrock/app/mst/mst_enactor.cuh>
#include <gunrock/app/mst/mst_problem.cuh>
#include <gunrock/app/mst/mst_functor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>
#include <moderngpu.cuh>

// CPU Prim's mst reference
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/prim_minimum_spanning_tree.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::mst;

/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/
bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
	 printf("\ntest_mst <graph type> <graph type args> [--device=<device_index>] "
        	"[--instrumented] [--quick] "
        	"[--v]\n"
        	"\n"
        	"Graph types and args:\n"
        	"  market [<file>]\n"
        	"    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
       		"    edges from stdin (or from the optionally-specified file).\n"
        	"  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        	"  --instrumented If set then kernels keep track of queue-search_depth\n"
        	"  and barrier duty (a relative indicator of load imbalance.)\n"
        	"  --quick If set will skip the CPU validation code.\n"
        	);
}

/**
 * @brief Displays the MST result
 *
 */
template<typename Value, typename SizeT>
void DisplaySolution()
{	/* 
	printf("\nVertex List (row_offsets):\n");
        for (SizeT node = 0;
        	node < csr.nodes;
                node++){
                	util::PrintValue(csr.row_offsets[node]);
                        printf(" ");
                }
	printf("\n");
        printf("\nEdge List (col_indices):\n");
        for (SizeT edge = 0;
        	edge < csr.edges;
                edge++){
                        util::PrintValue(csr.column_indices[edge]);
                        printf(" ");
                }
	printf("\n");		
	*/
}

 
/******************************************************************************
 * MST Testing Routines
 *****************************************************************************/
/**
 * @brief A simple CPU-based reference MST implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 */
template<
	typename VertexId,
	typename Value,
	typename SizeT>
void SimpleReferenceMST(
    	Value	*weights,
	const Csr<VertexId, Value, SizeT> &graph)
{
    	//Preparation
	using namespace boost;
	typedef adjacency_list < vecS, vecS, undirectedS,
		property < vertex_distance_t, int >, property < edge_weight_t, int > > Graph;
    	typedef std::pair < int, int > E;
	int num_nodes = graph.nodes;
	int num_edges = graph.edges;
	E *edge_pairs = new E[num_edges];
	int idx = 0;
	printf("node %d edge %d\n", num_nodes, num_edges);
	
	for (int i = 0; i < num_nodes; ++i)
	{
		for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
		{
			edge_pairs[idx++] = std::make_pair(i, graph.column_indices[j]);
		}

	}
   	 /*Graph g(num_nodes);
 	property_map<Graph, edge_weight_t>::type weightmap = get(edge_weight, g); 
 	for (std::size_t j = 0; j < sizeof(edge_pairs) / sizeof(E); ++j) {
 		printf("%d, %d\n", edge_pairs[j].first, edge_pairs[j].second);
 	graph_traits<Graph>::edge_descriptor e; bool inserted;
 	tie(e, inserted) = add_edge(edge_pairs[j].first, edge_pairs[j].second, g);
 	weightmap[e] = weights[j];
 	}*/
 	Graph g(edge_pairs, edge_pairs + num_edges, weights, num_nodes);
 	property_map<Graph, edge_weight_t>::type weightmap = get(edge_weight, g);
 	std::vector < graph_traits < Graph >::vertex_descriptor >
 		p(num_vertices(g));
 
 	typedef graph_traits<Graph>::edge_iterator edge_iterator;
 
 	std::pair<edge_iterator, edge_iterator> ei = edges(g);
 	for(edge_iterator edge_iter = ei.first; edge_iter != ei.second; ++edge_iter) 
	{
 		std::cout << "(" << source(*edge_iter, g) << ", " << target(*edge_iter, g) << ")\n";
      	}
	
    	//compute MST
    	CpuTimer cpu_timer;
    	cpu_timer.Start();
	prim_minimum_spanning_tree(g, &p[0]);	

    	cpu_timer.Stop();
    	float elapsed = cpu_timer.ElapsedMillis();

    	printf("CPU MST finished in %lf msec.\n", elapsed);
	
	for (std::size_t i = 0; i != p.size(); ++i)
		if (p[i] != i)
			std::cout << "parent[" << i << "] = " << p[i] << std::endl;
		else
			std::cout << "parent[" << i << "] = no parent" << std::endl;

}

/**
 * @brief Run MST tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    	typename VertexId,
    	typename Value,
    	typename SizeT,
    	bool INSTRUMENT>
void RunTests(
    	const Csr<VertexId, Value, SizeT> &graph,
    	int max_grid_size,
    	int num_gpus,
    	mgpu::CudaContext& context)
{
	/* Define the problem data structure for graph primitive */
    	typedef MSTProblem<
        VertexId,
        SizeT,
        Value,
	true> Problem;
	
	/* INSTRUMENT specifies whether we want to keep such statistical data */
    	/* Allocate MST enactor map */
    	MSTEnactor<INSTRUMENT> mst_enactor(g_verbose);

    	/* Allocate problem on GPU */
    	/* Create a pointer of the MSTProblem type */
	Problem *mst_problem = new Problem;
    	/* Copy data from CPU to GPU */
	/* Initialize data members in DataSlice */
	util::GRError(mst_problem->Init(
        	g_stream_from_host,
                graph,
                num_gpus), "Problem MST Initialization Failed", __FILE__, __LINE__);

    	// Perform MST
    	GpuTimer gpu_timer; /* Record the kernel running time */
	/* Reset values in DataSlice */
        util::GRError(mst_problem->Reset(mst_enactor.GetFrontierType()), 
		"MST Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(mst_enactor.template Enact<Problem>(context, mst_problem, max_grid_size), 
		"MST Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();

        float elapsed = gpu_timer.ElapsedMillis();

        /* Copy out results back to CPU from GPU using Extract */
        // TODO: write the extract function
        // util::GRError(csr_problem->Extract(h_result), 
	//	"MST Problem Data Extraction Failed", __FILE__, __LINE__);

        /* Verify the result using CompareResults() */
	// SimpleReferenceMST(graph.edge_values, graph);        

	/* Display solution*/
	//DisplaySolution()
		
        /* Cleanup */
        if (mst_problem) delete mst_problem;

        hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
	typename VertexId,
    	typename Value,
    	typename SizeT>
void RunTests(
    	Csr<VertexId, Value, SizeT> &graph,
    	CommandLineArgs &args,
    	mgpu::CudaContext& context)
{
    	bool instrumented = false;
    	int max_grid_size = 0;            
    	int num_gpus = 1;            

    	instrumented = args.CheckCmdLineFlag("instrumented");

    	g_quick = args.CheckCmdLineFlag("quick");
    	g_verbose = args.CheckCmdLineFlag("v");

    	if (instrumented) {
        RunTests<VertexId, Value, SizeT, true>(
                        graph,
                        max_grid_size,
                        num_gpus,
                        context);
    	} else {
        	RunTests<VertexId, Value, SizeT, false>(
                        graph,
                        max_grid_size,
                        num_gpus,
                        context);
    	}
}



/******************************************************************************
* Main
******************************************************************************/

int main(int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	//DeviceInit(args);
	//hipSetDeviceFlags(hipDeviceMapHost);
	int dev = 0;
    	args.GetCmdLineArgument("device", dev);
   	mgpu::ContextPtr context = mgpu::CreateCudaDevice(dev);
	//srand(0);			// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = true;

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		/* Matrix-market coordinate-formatted graph file */

		typedef int VertexId;	// Use as the node identifier type
		typedef int Value;	// Use as the value type
		typedef int SizeT;	// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);	
		/* Default value for stream_from_host is false */

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		/* BuildMarketGraph() reads a mtx file into CSR data structure */
		/* Template argumet = true because the graph has edge weights */
		if (graphio::BuildMarketGraph<true>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) // no inverse graph
		{
			return 1;
		}
			
		csr.DisplayGraph();
		/*	
		for (int i = 0; i < csr.edges; ++i)
         	{
             		printf("%d ", csr.edge_values[i]);
         	}
         	printf("\n");

		Csr<VertexId, Value, SizeT> csr2(false);
		graphio::BuildMarketGraph<true>(
		market_filename,
		csr2,
		false,
		false);

		csr2.DisplayGraph();
		SimpleReferenceMST(csr2.edge_values, csr2);
		*/
		// Run tests
		RunTests(csr, args, *context);
	
	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

	return 0;
}
