// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_mst.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <utility>
#include <iostream>
#include <cstdlib>
#include <algorithm>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// MST includes
#include <gunrock/app/mst/mst_enactor.cuh>
#include <gunrock/app/mst/mst_problem.cuh>
#include <gunrock/app/mst/mst_functor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>
#include <moderngpu.cuh>

// CPU Prim's mst reference
#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/prim_minimum_spanning_tree.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::mst;

/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/
bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
	printf("\ntest_mst <graph type> <graph type args> [--device=<device_index>] "
		"[--instrumented] [--quick] "
		"[--v]\n"
		"\n"
		"Graph types and args:\n"
		"  market [<file>]\n"
		"    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
		"    edges from stdin (or from the optionally-specified file).\n"
		"  --device=<device_index>  Set GPU device for running the graph primitive.\n"
		"  --instrumented If set then kernels keep track of queue-search_depth\n"
		"  and barrier duty (a relative indicator of load imbalance.)\n"
		"  --quick If set will skip the CPU validation code.\n"
		);
}

/**
 * @brief Displays the MST result
 *
 */
template<typename Value, typename SizeT>
void DisplaySolution()
{	
	/* 
	printf("\nVertex List (row_offsets):\n");
        for (SizeT node = 0;
      	node < csr.nodes;
        node++){
          	util::PrintValue(csr.row_offsets[node]);
            printf(" ");
        }
	printf("\n");
    printf("\nEdge List (col_indices):\n");
    for (SizeT edge = 0;
     	edge < csr.edges;
        edge++){
                    util::PrintValue(csr.column_indices[edge]);
                    printf(" ");
                }
	printf("\n");		
	*/
}

/**
 * @brief Comparison for the MST result
 *
 */
int compareResults()
{
	printf(" Comparing results ...\n");
	return 0;
}

/******************************************************************************
 * MST Testing Routines
 *****************************************************************************/
/**
 * @brief A simple CPU-based reference MST implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 */
template<
	typename VertexId,
	typename Value,
	typename SizeT>
void SimpleReferenceMST(
    Value	*weights,
	const Csr<VertexId, Value, SizeT> &graph)
{
    //Preparation
	using namespace boost;
	typedef adjacency_list < vecS, vecS, undirectedS,
		property<vertex_distance_t, int>, property < edge_weight_t, int > > Graph;
    typedef std::pair < int, int >E;
	const int num_nodes = graph.nodes;
	const int num_edges = graph.edges;
	
	printf(" Number of nodes: %4d, Number of edges: %4d\n", num_nodes, num_edges);
	E *edge_pairs = new E[num_edges];
	int idx = 0;
    
	for (int i = 0; i < num_nodes; ++i)
	{
		for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
		{
			edge_pairs[idx++] = std::make_pair(i, graph.column_indices[j]);
        } 
    }

    // original total weights
    int weight_sum = 0;
    for (int edgeIter = 0; edgeIter < num_edges; ++edgeIter)
    {
        weight_sum += weights[edgeIter];
    }
    printf(" Original Total Weights: %d\n", weight_sum);
       
    /*
	Graph g(num_nodes);
 	property_map<Graph, edge_weight_t>::type weightmap = get(edge_weight, g); 
 	for (std::size_t j = 0; j < sizeof(edge_pairs) / sizeof(E); ++j) 
	{
 		printf("%d, %d\n", edge_pairs[j].first, edge_pairs[j].second);
 		graph_traits<Graph>::edge_descriptor e; bool inserted;
 		tie(e, inserted) = add_edge(edge_pairs[j].first, edge_pairs[j].second, g);
 		weightmap[e] = weights[j];
 	}*/
 	
	Graph g(edge_pairs, edge_pairs + num_edges, weights, num_nodes);
 		property_map<Graph, edge_weight_t>::type weightmap = get(edge_weight, g);
 	std::vector < graph_traits < Graph >::vertex_descriptor >
 		p(num_vertices(g));
 
 	typedef graph_traits<Graph>::edge_iterator edge_iterator;
 
 	std::pair<edge_iterator, edge_iterator> ei = edges(g);
	
    /*
    // display graph
	for (edge_iterator edge_iter = ei.first; edge_iter != ei.second; ++edge_iter) 
	{
 		std::cout << "(" << source(*edge_iter, g) << ", " << target(*edge_iter, g) << ")\n";
    }
    */
    
    // Compute MST using CPU
    CpuTimer cpu_timer; // record the kernel running time  	
	
	cpu_timer.Start();
	prim_minimum_spanning_tree(g, &p[0]);
	cpu_timer.Stop();
    	
	float elapsed_cpu = cpu_timer.ElapsedMillis();
	
    printf(" CPU MST finished in %lf msec.\n", elapsed_cpu);
	
    // display graph results 	
    int final_edge_len = 0;
    for (std::size_t i = 0; i != p.size(); ++i)
	{
		if (p[i] != i)
        {
            std::cout << "parent[" << i << "] = " << p[i] << std::endl;
            final_edge_len += 1;
        }
        else
			std::cout << "parent[" << i << "] = no parent" << std::endl;
	}
	
	printf(" Number of edges selected - %d\n", p.size());
    // int total_weights_cpu = 0;
    // std::cout << "final edge len = " << final_edge_len << std::endl;
    // std::cout << "total weights cpu = " << total_weights_cpu << std::endl; 
    
    printf(" --- CPU MST Complete ---\n");
}

/**
 * @brief Run MST tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    int max_grid_size,
    int num_gpus,
    mgpu::CudaContext& context)
{
	/* Define the problem data structure for graph primitive */
    typedef MSTProblem<
        VertexId,
        SizeT,
        Value,
	true> Problem;
	
	/* INSTRUMENT specifies whether we want to keep such statistical data */
    // Allocate MST enactor map 
    MSTEnactor<INSTRUMENT> mst_enactor(g_verbose);

    /* Allocate problem on GPU */
    // Create a pointer of the MSTProblem type 
	Problem *mst_problem = new Problem;
    	
	/* Copy data from CPU to GPU */
	// Initialize data members in DataSlice 
	util::GRError(mst_problem->Init(
        g_stream_from_host,
        graph,
        num_gpus), "Problem MST Initialization Failed", __FILE__, __LINE__);

    // Perform MST
    GpuTimer gpu_timer; // Record the kernel running time 
	
	/* Reset values in DataSlice */
    util::GRError(mst_problem->Reset(mst_enactor.GetFrontierType()), 
    	"MST Problem Data Reset Failed", __FILE__, __LINE__);
    
    gpu_timer.Start();
    util::GRError(mst_enactor.template Enact<Problem>(context, mst_problem, max_grid_size), 
	    "MST Problem Enact Failed", __FILE__, __LINE__);
    gpu_timer.Stop();

    float elapsed_gpu = gpu_timer.ElapsedMillis();
	printf(" GPU MST finished in %lf msec.\n", elapsed_gpu);
	
    /* Copy out results back to CPU from GPU using Extract */
    // TODO: write the extract function
    // util::GRError(csr_problem->Extract(h_result), 
	//	"MST Problem Data Extraction Failed", __FILE__, __LINE__);

	/* Display solution*/
	// DisplaySolution()
		
    /* Cleanup */
    if (mst_problem) delete mst_problem;

    hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
	typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args,
    mgpu::CudaContext& context)
{
    bool instrumented = false;
    int max_grid_size = 0;            
    int num_gpus = 1;            

    instrumented = args.CheckCmdLineFlag("instrumented");
    
    g_quick = args.CheckCmdLineFlag("quick");
    g_verbose = args.CheckCmdLineFlag("v");

    if (instrumented) 
	{
        RunTests<VertexId, Value, SizeT, true>(
            graph,
            max_grid_size,
            num_gpus,
            context);
    } 
    else 
    {
        RunTests<VertexId, Value, SizeT, false>(
            graph,
            max_grid_size,
            num_gpus,
            context);
    }
}



/******************************************************************************
* Main
******************************************************************************/

int main(int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) 
	{
		Usage();
		return 1;
	}

	//DeviceInit(args);
	//hipSetDeviceFlags(hipDeviceMapHost);
	int dev = 0;
   	args.GetCmdLineArgument("device", dev);
   	mgpu::ContextPtr context = mgpu::CreateCudaDevice(dev);
	//srand(0);			// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = true;

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) 
	{
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") 
	{

		/* Matrix-market coordinate-formatted graph file */

		typedef int VertexId;	// Use as the node identifier type
		typedef int Value;	// Use as the value type
		typedef int SizeT;	// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);	
		
		/* Default value for stream_from_host is false */
		if (graph_args < 1) 
		{ 
			Usage(); 
			return 1; 
		}
	
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		
		/* BuildMarketGraph() reads a mtx file into CSR data structure */
		// Template argumet = true because the graph has edge weights 
		if (graphio::BuildMarketGraph<true>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) // no inverse graph
		{
			return 1;
		}
		
		// display graph	
		csr.DisplayGraph();
		
		// run gpu tests
		RunTests(csr, args, *context);
		
		// run cpu reference test
		printf("\n --- CPU Reference Test ---\n");
		// build a directed graph required by cpu reference computing
		Csr<VertexId, Value, SizeT> csr2(false);
        graphio::BuildMarketGraph<true>(
            market_filename,
            csr2,
            true,
            false);
        csr2.DisplayGraph();
		
		SimpleReferenceMST(csr2.edge_values, csr2);
	
		// verify results using compareResults() function 
		// int result = compareResults();
		// printf(" Verifying results ... %s\n", (result == 1) ? "Success!" : "Failed!");
	}
	else 
	{
		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;
	}

	return 0;
}

/* end */
