// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_dc.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <utility>
#include <iostream>
#include <cstdlib>
#include <algorithm>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// Degree Centrality includes
#include <gunrock/app/dc/dc_enactor.cuh>
#include <gunrock/app/dc/dc_problem.cuh>
//#include <gunrock/app/dc/dc_functor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>
#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::dc;

/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/
bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
	printf("\ntest_top <graph type> <graph type args> [--device=<device_index>] "
		"[--instrumented] [--quick] "
		"[--v]\n"
		"\n"
		"Graph types and args:\n"
		"  market [<file>]\n"
		"    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
		"    edges from stdin (or from the optionally-specified file).\n"
		"  --device=<device_index>  Set GPU device for running the graph primitive.\n"
		"  --instrumented If set then kernels keep track of queue-search_depth\n"
		"  and barrier duty (a relative indicator of load imbalance.)\n"
		"  --quick If set will skip the CPU validation code.\n"
		);
}

/**
 * @brief Displays the top result
 *
 */
template<typename VertexId, typename Value, typename SizeT>
void DisplaySolution(VertexId *h_node_id, Value *h_degrees, SizeT num_nodes)
{
	// only display top K degrees
	if (num_nodes > 100)
	{
		num_nodes = 100;
	}
	printf("\n-----> Top %d degree nodes: \n", num_nodes);
	printf("[    node_id    |   num_degree  ]\n");
	for (int i = 0; i < num_nodes; ++i)
	{
		printf("%16d %16d\n", h_node_id[i], h_degrees[i]);
	}
}

/**
 * @brief Comparison for the Degree Centrality result
 *
 */
int compareResults()
{
	printf(" Comparing results ...\n");
	return 0;
}

/******************************************************************************
 * Degree Centrality Testing Routines
 *****************************************************************************/
/**
 * @brief A simple CPU-based reference DC implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 */
template<
	typename VertexId,
	typename Value,
	typename SizeT>
void SimpleReferenceDC(
    Value	*weights,
	const Csr<VertexId, Value, SizeT> &graph)
{
    // Preparation

    // Compute DC using CPU
    CpuTimer cpu_timer; // record the kernel running time  	
	
	cpu_timer.Start();
	
	cpu_timer.Stop();
    	
	float elapsed_cpu = cpu_timer.ElapsedMillis();
	
    printf(" CPU Degree Centrality finished in %lf msec.\n", elapsed_cpu);
    printf(" --- CPU Degree Centrality Complete ---\n");
}

/**
 * @brief Run DC tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    int max_grid_size,
    int num_gpus,
    mgpu::CudaContext& context)
{
	/* Define the problem data structure for graph primitive */
    typedef DCProblem<
        VertexId,
        SizeT,
        Value> Problem;
	
	/* INSTRUMENT specifies whether we want to keep such statistical data */
    // Allocate DC enactor map 
    DCEnactor<INSTRUMENT> dc_enactor(g_verbose);

    /* Allocate problem on GPU */
    // Create a pointer of the DCProblem type 
	Problem *dc_problem = new Problem;
    	
	/* Copy data from CPU to GPU */
	// Initialize data members in DataSlice 
	util::GRError(dc_problem->Init(
        g_stream_from_host,
        graph,
        num_gpus), "Problem DC Initialization Failed", __FILE__, __LINE__);

    // Perform Degree Centrality
    GpuTimer gpu_timer; // Record the kernel running time 
	
	/* Reset values in DataSlice */
    util::GRError(dc_problem->Reset(dc_enactor.GetFrontierType()), 
    	"DC Problem Data Reset Failed", __FILE__, __LINE__);
    
    gpu_timer.Start();
    util::GRError(dc_enactor.template Enact<Problem>(context, dc_problem, max_grid_size), 
			"DC Problem Enact Failed", __FILE__, __LINE__);
	gpu_timer.Stop();

	float elapsed_gpu = gpu_timer.ElapsedMillis();
	printf(" GPU Degree Centrality finished in %lf msec.\n", elapsed_gpu);
	
	// Copy out results back to CPU from GPU using Extract 
    VertexId	*h_node_id = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);
	Value 		*h_degrees = (Value*)malloc(sizeof(Value) * graph.nodes);
	
	util::GRError(dc_problem->Extract(h_node_id, h_degrees), 
		"DC Problem Data Extraction Failed", __FILE__, __LINE__);

	// Display solution
	DisplaySolution(h_node_id, h_degrees, graph.nodes);
		
    // Cleanup
    if (dc_problem) delete dc_problem;

    hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
	typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args,
    mgpu::CudaContext& context)
{
    bool instrumented = false;
    int max_grid_size = 0;            
    int num_gpus = 1;            

    instrumented = args.CheckCmdLineFlag("instrumented");
    
    g_quick = args.CheckCmdLineFlag("quick");
    g_verbose = args.CheckCmdLineFlag("v");

    if (instrumented) 
	{
        RunTests<VertexId, Value, SizeT, true>(
            graph,
            max_grid_size,
            num_gpus,
            context);
    } 
    else 
    {
        RunTests<VertexId, Value, SizeT, false>(
            graph,
            max_grid_size,
            num_gpus,
            context);
    }
}



/******************************************************************************
* Main
******************************************************************************/

int main(int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) 
	{
		Usage();
		return 1;
	}

	//DeviceInit(args);
	//hipSetDeviceFlags(hipDeviceMapHost);
	int dev = 0;
   	args.GetCmdLineArgument("device", dev);
   	mgpu::ContextPtr context = mgpu::CreateCudaDevice(dev);
	//srand(0);			// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = false;

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) 
	{
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") 
	{

		/* Matrix-market coordinate-formatted graph file */

		typedef int VertexId;	// Use as the node identifier type
		typedef int Value;		// Use as the value type
		typedef int SizeT;		// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);	
		
		/* Default value for stream_from_host is false */
		if (graph_args < 1) 
		{ 
			Usage(); 
			return 1; 
		}
	
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		
		/* BuildMarketGraph() reads a mtx file into CSR data structure */
		// Template argumet = true because the graph has edge weights 
		if (graphio::BuildMarketGraph<true>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) // no inverse graph
		{
			return 1;
		}

		// display graph	
		// csr.DisplayGraph();
		
		// run gpu tests
		RunTests(csr, args, *context);
	
	}
	else 
	{
		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;
	}

	return 0;
}

/* end */
