// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_tcsp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/tc/tc_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error metcage(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use float as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        bool quick   = parameters.Get<bool>("quick");
        bool quiet   = parameters.Get<bool>("quiet");
        int num_runs = parameters.Get<int>("num-runs");

        SizeT nodes = graph.nodes;
        VertexT *ref_tc_counts = new VertexT[nodes];
        if (!quick) {
            util::PrintMsg("__________________________", !quiet);

            float elapsed = app::tc::CPU_Reference(
                parameters,
                graph.csr(),
                ref_tc_counts
            );

            util::PrintMsg("__________________________\nRun CPU Reference Avg. in "
                + std::to_string(num_runs) + " iterations elapsed: "
                + std::to_string(elapsed)
                + " ms", !quiet);
        }

        std::vector<std::string> switches{"advance-mode"};
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [ref_tc_counts](util::Parameters &parameters, GraphT &graph)
            {
                return app::tc::RunTests(parameters, graph, ref_tc_counts);
            }));

        if (ref_tc_counts != NULL)
        {
            delete[] ref_tc_counts; ref_tc_counts = NULL;
        }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Triangle Counting");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::tc::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    return app::Switch_Types<
        app::VERTEXT_U32B | //app::VERTEXT_U64B |
        app::SIZET_U32B | //app::SIZET_U64B |
        app::VALUET_F64B | app::UNDIRECTED | app::DIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
