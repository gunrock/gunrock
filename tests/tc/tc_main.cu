// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/global_indicator/tc/tc_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use float as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph; // graph we process on

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());
        //GUARD_CU(graph.csr().Display());

        VertexT  *ref_communities = NULL;
        bool quick = parameters.Get<bool>("quick");
        // compute reference CPU TC solution
/*        if (!quick)
        {
            bool quiet = parameters.Get<bool>("quiet");
            std::string validation = parameters.Get<std::string>("validation");
            util::PrintMsg("Computing reference value ...", !quiet);

            SizeT nodes = graph.nodes;
            ref_communities = new VertexT[nodes];
            //int num_runs = parameters.Get<int>("omp-runs");
            //for (int i = 0; i < num_runs; i++)
            {
                int i = 0;
                util::PrintMsg("__________________________", !quiet);
                float elapsed = app::louvain::CPU_Reference(
                    parameters, graph.csr(), ref_communities);
                util::PrintMsg("--------------------------\nRun "
                    + std::to_string(i) + " elapsed: "
                    + std::to_string(elapsed)
                    + " ms, q = " + std::to_string(app::louvain::Get_Modularity(
                        graph, ref_communities))
                    , !quiet);
            }
        }*/

        std::vector<std::string> switches{"advance-mode"};
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [](util::Parameters &parameters, GraphT &graph)
            {
                bool quiet = parameters.Get<bool>("quiet");
                //bool quick = parameters.Get<bool>("quick");

                return app::tc::RunTests(parameters, graph);
            }));

        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Louvain (community detection)");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::tc::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    return app::Switch_Types<
        app::VERTEXT_U32B | //app::VERTEXT_U64B |
        app::SIZET_U32B | //app::SIZET_U64B |
        app::VALUET_F64B | app::DIRECTED | app::UNDIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
