// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_proj.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/proj/proj_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

namespace APP_NAMESPACE = app::proj;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        // CLI parameters
        bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");

        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        ValueT *ref_projections;

        if (!quick) {
            ref_projections = new ValueT[graph.nodes * graph.nodes];

            // If not in `quick` mode, compute CPU reference implementation
            util::PrintMsg("__________________________", !quiet);

            float elapsed = app::proj::CPU_Reference(
                graph.csr(),
                ref_projections,
                quiet);

            util::PrintMsg("--------------------------\n Elapsed: "
                + std::to_string(elapsed), !quiet);
        }

        std::vector<std::string> switches{"advance-mode"};
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [
                ref_projections
            ](util::Parameters &parameters, GraphT &graph)
            {
                return app::proj::RunTests(parameters, graph, ref_projections, util::DEVICE);
            }));

        if (!quick) {
            delete[] ref_projections; ref_projections = NULL;
        }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test graph_projections");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::proj::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    return app::Switch_Types<
        app::VERTEXT_U32B | app::VERTEXT_U64B |
        app::SIZET_U32B | app::SIZET_U64B |
        app::VALUET_F32B | app::DIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
