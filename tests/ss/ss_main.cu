// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/ss/ss_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use float as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        bool quick   = parameters.Get<bool>("quick");
        bool quiet   = parameters.Get<bool>("quiet");
        int num_runs = parameters.Get<int>("num-runs");

        VertexT *ref_scan_stats = NULL;
        if (!quick) {
            SizeT nodes = graph.nodes;
            ref_scan_stats = new VertexT[nodes];

            // for(int i = 0; i < num_runs; i++) {
                util::PrintMsg("__________________________", !quiet);

                float elapsed = app::ss::CPU_Reference(
                    parameters,
                    graph.csr(),
                    ref_scan_stats
                );

                util::PrintMsg("__________________________\nRun "
                    + std::to_string(0) + " elapsed: "
                    + std::to_string(elapsed)
                    + " ms", !quiet);
            // }
        }
//        return retval;

        std::vector<std::string> switches{"advance-mode"};
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [](util::Parameters &parameters, GraphT &graph)
            {
                bool quiet = parameters.Get<bool>("quiet");
                //bool quick = parameters.Get<bool>("quick");
                int num_runs = parameters.Get<int>("omp-runs");
                util::PrintMsg("num_runs: " + std::to_string(num_runs));
                std::string validation = parameters.Get<std::string>("validation");
/*                if (num_runs > 0)
                {
                    VertexT *omp_communities = new VertexT[graph.nodes];
                    for (int i = 0; i < num_runs; i++)
                    {
                        util::PrintMsg("__________________________", !quiet);
                        float elapsed = app::ss::OMP_Reference(
                            parameters, graph.csr(), omp_communities);
                        util::PrintMsg("--------------------------", !quiet);

                        if (validation == "each")
                        {
                            util::PrintMsg("Run " + std::to_string(i) + " elapsed: "
                                + std::to_string(elapsed) + " ms", !quiet);

                            app::ss::Validate_Results(parameters, graph,
                                omp_communities, ref_scan_stats);
                        } else {
                            util::PrintMsg("Run " + std::to_string(i) + " elapsed: "
                                + std::to_string(elapsed) + " ms, q = "
                                + std::to_string(app::ss::Get_Modularity(
                                    graph, omp_communities)), !quiet);
                        }
                    }
                    if (validation == "last")
                        app::ss::Validate_Results(parameters, graph,
                            omp_communities, ref_scan_stats);

                    if (ref_scan_stats == NULL)
                        ref_scan_stats = omp_communities;
                    else
                    {
                        delete[] omp_communities; omp_communities = NULL;
                    }
                }*/

                return app::ss::RunTests(parameters, graph);
            }));

        if (ref_scan_stats != NULL)
        {
            delete[] ref_scan_stats; ref_scan_stats = NULL;
        }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Scan Statistics");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::ss::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    return app::Switch_Types<
        app::VERTEXT_U32B | //app::VERTEXT_U64B |
        app::SIZET_U32B | //app::SIZET_U64B |
        app::VALUET_F64B | app::UNDIRECTED | app::DIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
