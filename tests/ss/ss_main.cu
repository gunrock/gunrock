// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/ss/ss_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use float as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        bool quick   = parameters.Get<bool>("quick");
        bool quiet   = parameters.Get<bool>("quiet");
        int num_runs = parameters.Get<int>("num-runs");

        SizeT nodes = graph.nodes;
        ValueT *ref_scan_stats = new ValueT[nodes];
        if (!quick) {
            util::PrintMsg("__________________________", !quiet);

            float elapsed = app::ss::CPU_Reference(
                parameters,
                graph.csr(),
                ref_scan_stats
            );

            util::PrintMsg("__________________________\nRun "
                + std::to_string(0) + " elapsed: "
                + std::to_string(elapsed)
                + " ms", !quiet);
        }

        std::vector<std::string> switches{"advance-mode"};
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [ref_scan_stats](util::Parameters &parameters, GraphT &graph)
            {
                return app::ss::RunTests(parameters, graph, ref_scan_stats);
            }));

        if (ref_scan_stats != NULL)
        {
            delete[] ref_scan_stats; ref_scan_stats = NULL;
        }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Scan Statistics");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::ss::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    return app::Switch_Types<
        app::VERTEXT_U32B | //app::VERTEXT_U64B |
        app::SIZET_U32B | //app::SIZET_U64B |
        app::VALUET_F64B | app::UNDIRECTED | app::DIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
