#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_mf.cu
 *
 * @brief Simple test driver program for max-flow algorithm.
 */

#include <gunrock/app/mf/mf_app.cu>
#include <gunrock/app/test_base.cuh>

#define debug_aml(a...)
//#define debug_aml(a...) {printf(a); printf("\n");}

using namespace gunrock;

/*****************************************************************************
* Main
*****************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT	  Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters, VertexT v, SizeT s, 
	    ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT, 
	  graph::HAS_EDGE_VALUES | graph::HAS_CSR> GraphT;
	typedef typename GraphT::CsrT CsrT;
        hipError_t retval = hipSuccess;
	bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");
	
	//
	// Load Graph
	//
        util::CpuTimer cpu_timer; cpu_timer.Start();
	debug_aml("Start Load Graph");
      
	bool undirected;
	parameters.Get("undirected", undirected);
	if (undirected){
	    debug_aml("graph is undirected");
	}else{
	    debug_aml("graph is directed");
	}

	GraphT d_graph;
	if (not undirected){
	    debug_aml("Load directed graph");
	    parameters.Set<int>("remove-duplicate-edges", false);
	    GUARD_CU(graphio::LoadGraph(parameters, d_graph));
	}

	debug_aml("Load undirected graph");
	GraphT u_graph;
	parameters.Set<int>("undirected", 1);
	parameters.Set<int>("remove-duplicate-edges", true);
        GUARD_CU(graphio::LoadGraph(parameters, u_graph));
	
	cpu_timer.Stop();

	parameters.Set("load-time", cpu_timer.ElapsedMillis());
	debug_aml("load-time is %lf",cpu_timer.ElapsedMillis());

	if (parameters.Get<VertexT>("source") == 
		util::PreDefinedValues<VertexT>::InvalidValue){
	    parameters.Set("source", 0);
	}
	if (parameters.Get<VertexT>("sink") == 
		util::PreDefinedValues<VertexT>::InvalidValue){
	    parameters.Set("sink", u_graph.nodes-1);
	}

	VertexT source = parameters.Get<VertexT>("source");
	VertexT sink = parameters.Get<VertexT>("sink");

	if (not undirected){
	    debug_aml("Directed graph:");
	    debug_aml("number of edges %d", d_graph.edges);
	    debug_aml("number of nodes %d", d_graph.nodes);
	}

	debug_aml("Undirected graph:");
	debug_aml("number of edges %d", u_graph.edges);
	debug_aml("number of nodes %d", u_graph.nodes);

	ValueT* flow_edge = (ValueT*)malloc(sizeof(ValueT)*u_graph.edges);
	SizeT* reverse	  = (SizeT*)malloc(sizeof(SizeT)*u_graph.edges);

	// Initialize reverse array.
	for (auto u = 0; u < u_graph.nodes; ++u)
	{
	    auto e_start = u_graph.CsrT::GetNeighborListOffset(u);
	    auto num_neighbors = u_graph.CsrT::GetNeighborListLength(u);
	    auto e_end = e_start + num_neighbors;
	    for (auto e = e_start; e < e_end; ++e)
	    {
		auto v = u_graph.CsrT::GetEdgeDest(e);
		auto f_start = u_graph.CsrT::GetNeighborListOffset(v);
		auto num_neighbors2 = u_graph.CsrT::GetNeighborListLength(v);
		auto f_end = f_start + num_neighbors2;
		for (auto f = f_start; f < f_end; ++f)
		{
		    auto z = u_graph.CsrT::GetEdgeDest(f);
		    if (z == u)
		    {
			reverse[e] = f;
			reverse[f] = e;
			break;
		    }
		}
	    }
	}

	if (not undirected){
	    // Correct capacity values on reverse edges
	    for (auto u = 0; u < u_graph.nodes; ++u)
	    {
		auto e_start = u_graph.CsrT::GetNeighborListOffset(u);
		auto num_neighbors = u_graph.CsrT::GetNeighborListLength(u);
		auto e_end = e_start + num_neighbors;
		debug_aml("vertex %d\nnumber of neighbors %d", u, 
			num_neighbors);
		for (auto e = e_start; e < e_end; ++e)
		{
		    u_graph.CsrT::edge_values[e] = (ValueT)0;
		    auto v = u_graph.CsrT::GetEdgeDest(e);
		    // Looking for edge u->v in directed graph
		    auto f_start = d_graph.CsrT::GetNeighborListOffset(u);
		    auto num_neighbors2 = 
			d_graph.CsrT::GetNeighborListLength(u);
		    auto f_end = f_start + num_neighbors2;
		    for (auto f = f_start; f < f_end; ++f)
		    {
			auto z = d_graph.CsrT::GetEdgeDest(f);
			if (z == v and d_graph.CsrT::edge_values[f] > 0)
			{
			    u_graph.CsrT::edge_values[e]  = 
				d_graph.CsrT::edge_values[f];
			    debug_aml("edge (%d, %d) cap = %lf\n", u, v, \
				    u_graph.CsrT::edge_values[e]);
			    break;
			}
		    }
		}
	    }
	}
/*
	ValueT** rGraph = (ValueT**)malloc(sizeof(ValueT*)*u_graph.nodes);
	for (auto x = 0; x < u_graph.nodes; ++x){
	    rGraph[x] = (ValueT*)malloc(sizeof(ValueT)*u_graph.nodes);
	    for (auto y = 0; y < u_graph.nodes; ++y){
		rGraph[x][y] = (ValueT)0;
	    }
	}
	for (auto x = 0; x < u_graph.nodes; ++x)
	{
	    auto e_start = u_graph.CsrT::GetNeighborListOffset(x);
	    auto num_neighbors = u_graph.CsrT::GetNeighborListLength(x);
	    auto e_end = e_start + num_neighbors;
	    for (auto e = e_start; e < e_end; ++e){
		auto y = u_graph.CsrT::GetEdgeDest(e);
		auto f = u_graph.CsrT::edge_values[e];
		rGraph[x][y] = f;
	    }
	}

	FILE * rgraph_file = fopen("rgraph_output", "w");
	printf("number of nodes %d\n", u_graph.nodes);
	fprintf(rgraph_file, "{");
	for (auto n = 0; n < u_graph.nodes; ++n){
	    fprintf(rgraph_file, "{");
	    for (auto m = 0; m < u_graph.nodes; ++m){
		fprintf(rgraph_file, "%.0lf", rGraph[n][m]);
		if (m == u_graph.nodes-1){
		    fprintf(rgraph_file, " ");
		}else{
		    fprintf(rgraph_file, ", ");
		}
	    }
	    fprintf(rgraph_file, "}\n");
	}
	fprintf(rgraph_file, "}");
	fclose(rgraph_file);
*/

	//
        // Compute reference CPU max flow algorithm.
	//
        ValueT max_flow;
	
	if (!quick) {
	    util::PrintMsg("______CPU reference algorithm______", true);
	    double elapsed = app::mf::CPU_Reference
	        (parameters, u_graph, source, sink, max_flow, reverse, flow_edge);
            util::PrintMsg("-----------------------------------\nElapsed: " + 
		std::to_string(elapsed) + " ms\n Max flow CPU = " +
		std::to_string(max_flow), true);
	}

        std::vector<std::string> switches{"advance-mode"};
	GUARD_CU(app::Switch_Parameters(parameters, u_graph, switches,
	[flow_edge, reverse](util::Parameters &parameters, GraphT &u_graph)
	{
	  debug_aml("go to RunTests");
	  return app::mf::RunTests(parameters, u_graph, reverse, flow_edge);
	}));

	// Clean up
	free(flow_edge);
	free(reverse);
	
        return retval;
    }
};

int main(int argc, char** argv)
{
    debug_aml("Main: start");
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test mf");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::mf::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());
    debug_aml("Main: parameters checked - ok");

    return app::Switch_Types<
        app::VERTEXT_U32B | 
        app::SIZET_U32B | 
        app::VALUET_F64B | 
	app::DIRECTED | app::UNDIRECTED >
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:

