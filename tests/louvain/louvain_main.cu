// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * louvain_main.cu
 *
 * @brief Simple test driver program for Louvain
 */

#include <gunrock/app/louvain/louvain_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use float as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph; // graph we process on

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());
        //GUARD_CU(graph.csr().Display());

        VertexT  *ref_communities = NULL;
        bool quick = parameters.Get<bool>("quick");
        // compute reference CPU Louvain solution
        if (!quick)
        {
            bool quiet = parameters.Get<bool>("quiet");
            std::string validation = parameters.Get<std::string>("validation");
            util::PrintMsg("Computing reference value ...", !quiet);

            SizeT nodes = graph.nodes;
            ref_communities = new VertexT[nodes];
            //int num_runs = parameters.Get<int>("omp-runs");
            //for (int i = 0; i < num_runs; i++)
            {
                int i = 0;
                util::PrintMsg("__________________________", !quiet);
                float elapsed = app::louvain::CPU_Reference(
                    parameters, graph.csr(), ref_communities);
                util::PrintMsg("--------------------------\nRun "
                    + std::to_string(i) + " elapsed: "
                    + std::to_string(elapsed)
                    + " ms, q = " + std::to_string(app::louvain::Get_Modularity(
                        graph, ref_communities))
                    , !quiet);
            }
        }

        std::vector<std::string> switches{"unify-segments", 
            "advance-mode","omp-threads", "1st-th", "neighborcomm-th"};
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [&ref_communities](util::Parameters &parameters, GraphT &graph)
            {
                bool quiet = parameters.Get<bool>("quiet");
                //bool quick = parameters.Get<bool>("quick");
                int num_runs = parameters.Get<int>("omp-runs");
                std::string validation = parameters.Get<std::string>("validation");
                if (num_runs > 0)
                {
                    VertexT *omp_communities = new VertexT[graph.nodes];
                    for (int i = 0; i < num_runs; i++)
                    {
                        util::PrintMsg("__________________________", !quiet);
                        float elapsed = app::louvain::OMP_Reference(
                            parameters, graph.csr(), omp_communities);
                        util::PrintMsg("--------------------------", !quiet);

                        if (validation == "each")
                        {
                            util::PrintMsg("Run " + std::to_string(i) + " elapsed: "
                                + std::to_string(elapsed) + " ms", !quiet);
                            
                            app::louvain::Validate_Results(parameters, graph, 
                                omp_communities, ref_communities);
                        } else {
                            util::PrintMsg("Run " + std::to_string(i) + " elapsed: "
                                + std::to_string(elapsed) + " ms, q = " 
                                + std::to_string(app::louvain::Get_Modularity(
                                    graph, omp_communities)), !quiet);
                        }
                    }
                    if (validation == "last")
                        app::louvain::Validate_Results(parameters, graph,
                            omp_communities, ref_communities);

                    if (ref_communities == NULL)
                        ref_communities = omp_communities;
                    else
                    {
                        delete[] omp_communities; omp_communities = NULL;
                    }
                }

                return app::louvain::RunTests(parameters, graph, ref_communities);
            }));

        if (ref_communities != NULL)
        {
            delete[] ref_communities; ref_communities = NULL;
        }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Louvain (community detection)");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::louvain::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    return app::Switch_Types<
        app::VERTEXT_U32B | //app::VERTEXT_U64B |
        app::SIZET_U32B | //app::SIZET_U64B |
        app::VALUET_F64B >
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
